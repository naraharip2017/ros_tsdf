#include "hip/hip_runtime.h"
#include "cuda/tsdf_handler.cuh"

const int threadsPerCudaBlock = 128;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define cdpErrchk(ans) { cdpAssert((ans), __FILE__, __LINE__); }
__device__ void cdpAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      printf("GPU kernel assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) assert(0);
   }
}

__global__
void printHashTableAndBlockHeap(HashTable * hashTable_d, BlockHeap * blockHeap_d){
  // HashEntry * hashEntries = hashTable_d->hashEntries;
  // for(size_t i=0;i<NUM_BUCKETS; ++i){
  //   printf("Bucket: %lu\n", (unsigned long)i);
  //   for(size_t it = 0; it<HASH_ENTRIES_PER_BUCKET; ++it){
  //     HashEntry hashEntry = hashEntries[it+i*HASH_ENTRIES_PER_BUCKET];
  //     Vector3f position = hashEntry.position;
  //     if (hashEntry.isFree()){
  //       printf("  Hash Entry with   Position: (N,N,N)   Offset: %d   Pointer: %d\n", hashEntry.offset, hashEntry.pointer);
  //     }
  //     else{
  //       printf("  Hash Entry with   Position: (%f,%f,%f)   Offset: %d   Pointer: %d\n", position(0), position(1), position(2), hashEntry.offset, hashEntry.pointer);
  //     }
  //   }
  //   printf("%s\n", "--------------------------------------------------------");
  // }

  // printf("Block Heap Free List: ");
  // int * freeBlocks = blockHeap_d->freeBlocks;
  // for(size_t i = 0; i<NUM_HEAP_BLOCKS; ++i){
  //   printf("%d  ", freeBlocks[i]);
  // }
  printf("\nCurrent Index: %d\n", blockHeap_d->currentIndex);
}

__global__
void printVoxelBlocksFromPoint(Vector3f * pointCloudVoxelBlocks_d, int * pointer_d){
  printf("List of Points: \n");
  for(int i=0;i<*pointer_d;++i){
    Vector3f point = pointCloudVoxelBlocks_d[i];
    printf("(%f, %f, %f)\n", point(0), point(1), point(2));
  }
}

__device__
size_t retrieveHashIndexFromPoint(Vector3f point){ //tested using int can get negatives
  return abs((((int)point(0)*PRIME_ONE) ^ ((int)point(1)*PRIME_TWO) ^ ((int)point(2)*PRIME_THREE)) % NUM_BUCKETS);
}

__device__ 
float FloorFun(float x, float scale){
  return floor(x*scale) / scale;
}

__device__
Vector3f GetVoxelBlockCenterFromPoint(Vector3f point){
  float scale = 1 / VOXEL_BLOCK_SIZE;
  Vector3f blockCenter;
  blockCenter(0) = FloorFun(point(0), scale) + HALF_VOXEL_BLOCK_SIZE;
  blockCenter(1) = FloorFun(point(1), scale) + HALF_VOXEL_BLOCK_SIZE;
  blockCenter(2) = FloorFun(point(2), scale) + HALF_VOXEL_BLOCK_SIZE;
  return blockCenter;
}

__device__
Vector3f GetVoxelCenterFromPoint(Vector3f point){
  float scale = 1 / VOXEL_SIZE;
  Vector3f voxelCenter;
  voxelCenter(0) = FloorFun(point(0), scale) + HALF_VOXEL_SIZE;
  voxelCenter(1) = FloorFun(point(1), scale) + HALF_VOXEL_SIZE;
  voxelCenter(2) = FloorFun(point(2), scale) + HALF_VOXEL_SIZE;
  return voxelCenter;
}

__device__
bool checkFloatingPointVectorsEqual(Vector3f A, Vector3f B, float epsilon){
  Vector3f diff = A-B;
  if((fabs(diff(0)) < epsilon) && (fabs(diff(1)) < epsilon) && (fabs(diff(2)) < epsilon))
    return true;

  return false;
}

 __global__
 void getVoxelBlocksForPoint(pcl::PointXYZ * points_d, Vector3f * pointCloudVoxelBlocks_d, int * pointer_d, Vector3f * origin_transformed_d, int * size_d){
  int threadIndex = (blockIdx.x*128 + threadIdx.x);
  //printf("size: %d\n", *size_d);
  if(threadIndex>=*size_d){
    return;
  }
  pcl::PointXYZ point_d = points_d[threadIndex];
  Vector3f u = *origin_transformed_d;
  // printf("transformation: (%f, %f, %f)\n", u(0), u(1), u(2));
  Vector3f point_d_vector(point_d.x, point_d.y, point_d.z);
  Vector3f v = point_d_vector - u; //direction
  // printf("V: (%f, %f, %f)\n", v(0), v(1), v(2));
  //equation of line is u+tv
  float vMag = sqrt(pow(v(0), 2) + pow(v(1),2) + pow(v(2), 2));
  Vector3f v_normalized = v / vMag;
  Vector3f truncation_start = point_d_vector - truncation_distance*v_normalized;
  // printf("Truncation start : (%f, %f, %f)\n", truncation_start(0), truncation_start(1), truncation_start(2));
  
  Vector3f truncation_end = point_d_vector + truncation_distance*v_normalized;  //get voxel block of this and then traverse through voxel blocks till it equals this one
  // printf("Truncation end : (%f, %f, %f)\n", truncation_end(0), truncation_end(1), truncation_end(2));

  float distance_tStart_origin = sqrt(pow(truncation_start(0) - u(0), 2) + pow(truncation_start(1) - u(1),2) + pow(truncation_start(2) - u(2), 2));
  float distance_tEnd_origin = sqrt(pow(truncation_end(0) - u(0), 2) + pow(truncation_end(1) - u(1),2) + pow(truncation_end(2) - u(2), 2));

  if(distance_tEnd_origin < distance_tStart_origin){
    Vector3f temp = truncation_start;
    truncation_start = truncation_end;
    truncation_end = temp;
  }

  Vector3f truncation_start_block = GetVoxelBlockCenterFromPoint(truncation_start);
  // printf("Truncation start Block: (%f, %f, %f), hashes to %lu\n", truncation_start_block(0), truncation_start_block(1), truncation_start_block(2), retrieveHash(truncation_start_block));
  // printf("point in size_t: %d, %d, %d\n", (int)truncation_start_block(0), (int)truncation_start_block(1), (int)truncation_start_block(2));
  Vector3f truncation_end_block = GetVoxelBlockCenterFromPoint(truncation_end);
  // printf("Truncation end Block: (%f, %f, %f), hashes to %lu\n", truncation_end_block(0), truncation_end_block(1), truncation_end_block(2), retrieveHash(truncation_end_block));
  // printf("point in size_t: %d, %d, %d\n", (int)truncation_end_block(0), (int)truncation_end_block(1), (int)truncation_end_block(2));
  float stepX = v(0) > 0 ? VOXEL_BLOCK_SIZE : -1 * VOXEL_BLOCK_SIZE;
  float stepY = v(1) > 0 ? VOXEL_BLOCK_SIZE : -1 * VOXEL_BLOCK_SIZE;
  float stepZ = v(2) > 0 ? VOXEL_BLOCK_SIZE : -1 * VOXEL_BLOCK_SIZE;
  float tMaxX = fabs(v(0) < 0 ? (truncation_start_block(0) - HALF_VOXEL_BLOCK_SIZE - u(0)) / v(0) : (truncation_start_block(0) + HALF_VOXEL_BLOCK_SIZE - u(0)) / v(0));
  float tMaxY = fabs(v(1) < 0 ? (truncation_start_block(1) - HALF_VOXEL_BLOCK_SIZE - u(1)) / v(1) : (truncation_start_block(1) + HALF_VOXEL_BLOCK_SIZE - u(1)) / v(1));
  float tMaxZ = fabs(v(2) < 0 ? (truncation_start_block(2) - HALF_VOXEL_BLOCK_SIZE - u(2)) / v(2) : (truncation_start_block(2) + HALF_VOXEL_BLOCK_SIZE - u(2)) / v(2));
  float tDeltaX = fabs(VOXEL_BLOCK_SIZE / v(0));
  float tDeltaY = fabs(VOXEL_BLOCK_SIZE / v(1));
  float tDeltaZ = fabs(VOXEL_BLOCK_SIZE / v(2));
  Vector3f currentBlock(truncation_start_block(0), truncation_start_block(1), truncation_start_block(2));

  while(!checkFloatingPointVectorsEqual(currentBlock, truncation_end_block, EPSILON)){
    //add current block to blocks in current frame list or whatever
    int pointCloudVoxelBlocksIndex = atomicAdd(&(*pointer_d), 1);
    pointCloudVoxelBlocks_d[pointCloudVoxelBlocksIndex] = currentBlock;
    // printf("Current Block: (%f, %f, %f), hashes to %lu\n", currentBlock(0), currentBlock(1), currentBlock(2), retrieveHash
    // (currentBlock));
    // printf("point in size_t: %d, %d, %d\n", (int)currentBlock(0), (int)currentBlock(1), (int)currentBlock(2));
    if(tMaxX < tMaxY){
      if(tMaxX < tMaxZ)
      {
        currentBlock(0) += stepX;
        tMaxX += tDeltaX;
      }
      else if(tMaxX > tMaxZ){
        currentBlock(2) += stepZ;
        tMaxZ += tDeltaZ;
      }
      else{
        currentBlock(0) += stepX;
        currentBlock(2) += stepZ;
        tMaxX += tDeltaX;
        tMaxZ += tDeltaZ;
      }
    }
    else if(tMaxX > tMaxY){
      if(tMaxY < tMaxZ){
        currentBlock(1) += stepY;
        tMaxY += tDeltaY;
      }
      else if(tMaxY > tMaxZ){
        currentBlock(2) += stepZ;
        tMaxZ += tDeltaZ;
      }
      else{
        currentBlock(1) += stepY;
        currentBlock(2) += stepZ;
        tMaxY += tDeltaY;
        tMaxZ += tDeltaZ;
      }
    }
    else{
      if(tMaxZ < tMaxX){
        currentBlock(2) += stepZ;
        tMaxZ += tDeltaZ;
      }
      else if(tMaxZ > tMaxX){
        currentBlock(0) += stepX;
        currentBlock(1) += stepY;
        tMaxX += tDeltaX;
        tMaxY += tDeltaY;
      }
      else{ //can remove equals statements if want to improve on performance
        currentBlock(0) += stepX;
        currentBlock(1) += stepY;
        currentBlock(2) += stepZ;
        tMaxX += tDeltaX;
        tMaxY += tDeltaY;
        tMaxZ += tDeltaZ;
      }
    } 
  }      
  
  int pointCloudVoxelBlocksIndex = atomicAdd(&(*pointer_d), 1);
  pointCloudVoxelBlocks_d[pointCloudVoxelBlocksIndex] = currentBlock;
  // printf("Current Block: (%f, %f, %f), hashes to %lu\n", currentBlock(0), currentBlock(1), currentBlock(2), retrieveHash(currentBlock));
  // printf("point in size_t: %d, %d, %d\n", (int)currentBlock(0), (int)currentBlock(1), (int)currentBlock(2));
  // printf("Cloud with Points: %f, %f, %f\n", points_d[threadIndex].x,points_d[threadIndex].y,points_d[threadIndex].z);
  return;
 }

 __global__
void allocateVoxelBlocks(Vector3f * points_d, HashTable * hashTable_d, BlockHeap * blockHeap_d, bool * unallocatedPoints_d, int * size_d, int * unallocatedPointsCount_d)
{
  
  int threadIndex = (blockIdx.x*threadsPerCudaBlock + threadIdx.x);
  if(threadIndex>=*size_d || (unallocatedPoints_d[threadIndex]==0)){
    return;
  }
  Vector3f point_d = points_d[threadIndex];
  size_t bucketIndex = retrieveHashIndexFromPoint(point_d);
  size_t currentGlobalIndex = bucketIndex * HASH_ENTRIES_PER_BUCKET;
  //printf("Point: (%f, %f, %f), Index: %lu\n", point_d(0), point_d(1), point_d(2), bucketIndex);
  HashEntry * hashEntries = hashTable_d->hashEntries;
  bool blockNotAllocated = true;
  HashEntry hashEntry;
  for(size_t i=0; i<HASH_ENTRIES_PER_BUCKET; ++i){
    hashEntry = hashEntries[currentGlobalIndex+i];
    if(hashEntry.checkIsPositionEqual(point_d)){
      unallocatedPoints_d[threadIndex] = 0;
      atomicSub(unallocatedPointsCount_d, 1);
      blockNotAllocated = false;
      return; //todo: return reference to block
      //update this to just return
      //return
    }
  }

  //set currentGlobalIndex to last position in bucket to check linked list
  currentGlobalIndex+=HASH_ENTRIES_PER_BUCKET-1;

  //check linked list
  while(hashEntry.offset!=0){
    short offset = hashEntry.offset;
    currentGlobalIndex+=offset;
    if(currentGlobalIndex>=HASH_TABLE_SIZE){
      currentGlobalIndex %= HASH_TABLE_SIZE;
    }
    hashEntry = hashEntries[currentGlobalIndex];
    if(hashEntry.checkIsPositionEqual(point_d)){ //what to do if positions are 0,0,0 then every initial block will map to the point
      unallocatedPoints_d[threadIndex] = 0;
      atomicSub(unallocatedPointsCount_d, 1);
      blockNotAllocated = false; //update this to just return
      // printf("%s", "block allocated");
      return; //todo: return reference to block
      //return
    }
  }

  //can have a full checker boolean if true then skip checking the hashed bucket for writing

  //leads to divergent threads so break these up

  size_t insertCurrentGlobalIndex = bucketIndex * HASH_ENTRIES_PER_BUCKET;

  //allocate block
  if(blockNotAllocated){
    if(!atomicCAS(&hashTable_d->mutex[bucketIndex], 0, 1)){
        VoxelBlock * allocBlock = new VoxelBlock();
        bool notInserted = true;
        for(size_t i=0; i<HASH_ENTRIES_PER_BUCKET; ++i){
          HashEntry entry = hashEntries[insertCurrentGlobalIndex+i];
          if(entry.isFree()){ 
            int blockHeapFreeIndex = atomicAdd(&(blockHeap_d->currentIndex), 1);
            blockHeap_d->blocks[blockHeapFreeIndex] = *allocBlock;
            hipFree(allocBlock);
            HashEntry * allocBlockHashEntry = new HashEntry(point_d, blockHeapFreeIndex);
            hashEntries[insertCurrentGlobalIndex+i] = *allocBlockHashEntry;
            hipFree(allocBlockHashEntry);
            notInserted = false;
            unallocatedPoints_d[threadIndex] = 0;
            atomicSub(unallocatedPointsCount_d, 1);
            atomicExch(&hashTable_d->mutex[bucketIndex], 0);
            return;
          }
        }

        size_t insertBucketIndex = bucketIndex + 1;
        if(insertBucketIndex == NUM_BUCKETS){
          insertBucketIndex = 0;
        }

        bool haveLinkedListBucketLock = true;

        //check bucket of linked list end if different release hashbucket lock
        size_t endLinkedListBucket = currentGlobalIndex / HASH_ENTRIES_PER_BUCKET;
        if(endLinkedListBucket!=bucketIndex){
          atomicExch(&hashTable_d->mutex[bucketIndex], 0);
          haveLinkedListBucketLock = !atomicCAS(&hashTable_d->mutex[endLinkedListBucket], 0, 1);
        }

        if(haveLinkedListBucketLock){
                  //find position outside of current bucket
            while(notInserted){ //grab atomicCAS of linked list before looping for free spot
              //check offset of head linked list pointer
              if(!atomicCAS(&hashTable_d->mutex[insertBucketIndex], 0, 1)){
                insertCurrentGlobalIndex = insertBucketIndex * HASH_ENTRIES_PER_BUCKET;
                for(size_t i=0; i<HASH_ENTRIES_PER_BUCKET-1; ++i){
                  HashEntry entry = hashEntries[insertCurrentGlobalIndex+i];
                  //make this a method like entry.checkFree super unclean currently
                  if(entry.isFree() ){ //what to do if positions are 0,0,0 then every initial block will map to the point - set initial position to null in constructor
                    //set offset of last linked list node
                      int blockHeapFreeIndex = atomicAdd(&(blockHeap_d->currentIndex), 1);
                      blockHeap_d->blocks[blockHeapFreeIndex] = *allocBlock;
                      HashEntry * allocBlockHashEntry = new HashEntry(point_d, blockHeapFreeIndex);
                      size_t insertPos = insertCurrentGlobalIndex + i;
                      hashEntries[insertPos] = *allocBlockHashEntry;
                      hipFree(allocBlock);
                      hipFree(allocBlockHashEntry);
                      if(insertPos > currentGlobalIndex){
                        hashEntries[currentGlobalIndex].offset = insertPos - currentGlobalIndex;
                      }
                      else{
                        hashEntries[currentGlobalIndex].offset = HASH_TABLE_SIZE - currentGlobalIndex + insertPos;
                      }
                      notInserted = false;
                      unallocatedPoints_d[threadIndex] = 0;
                      atomicSub(unallocatedPointsCount_d, 1);
                    
                    break;
                  }
                }
                atomicExch(&hashTable_d->mutex[insertBucketIndex], 0);
              }
              insertBucketIndex++;
              if(insertBucketIndex == NUM_BUCKETS){
                insertBucketIndex = 0;
              }
              if(insertBucketIndex == bucketIndex){
                // unallocatedPoints_d[threadIndex] = 1;
                return;
              }
              //check if equals hashedbucket then break, only loop through table once then have to return point for next frame
            }
            atomicExch(&hashTable_d->mutex[endLinkedListBucket], 0);
      }
      else{
        // unallocatedPoints_d[threadIndex] = 1;
        return;
      }

      //free block here or we have another kernel in parallel reset all mutex
    }
    //printf("thread id: %d, mutex: %d\n", threadIdx.x, mutex);
    //determine which blocks are not inserted
    else{
      // unallocatedPoints_d[threadIndex] = 1;
    }
  }
  return;
}

__device__
size_t getLocalVoxelIndex(Vector3f diff){
  diff /= VOXEL_SIZE;
  return floor(diff(0)) + (floor(diff(1)) * VOXEL_PER_BLOCK) + (floor(diff(2)) * VOXEL_PER_BLOCK * VOXEL_PER_BLOCK);
}

__device__ 
int getBlockPositionForBlockCoordinates(Vector3f voxelBlockCoordinates, HashTable * hashTable_d){
  size_t bucketIndex = retrieveHashIndexFromPoint(voxelBlockCoordinates);
  size_t currentGlobalIndex = bucketIndex * HASH_ENTRIES_PER_BUCKET;
  HashEntry * hashEntries = hashTable_d->hashEntries;
  HashEntry hashEntry;
  for(size_t i=0; i<HASH_ENTRIES_PER_BUCKET; ++i){
    hashEntry = hashEntries[currentGlobalIndex+i];
    if(hashEntry.checkIsPositionEqual(voxelBlockCoordinates)){
      return hashEntry.pointer;
    }
  }

  currentGlobalIndex+=HASH_ENTRIES_PER_BUCKET-1;

  //check linked list
  while(hashEntry.offset!=0){
    short offset = hashEntry.offset;
    currentGlobalIndex+=offset;
    if(currentGlobalIndex>=HASH_TABLE_SIZE){
      currentGlobalIndex %= HASH_TABLE_SIZE;
    }
    hashEntry = hashEntries[currentGlobalIndex];
    if(hashEntry.checkIsPositionEqual(voxelBlockCoordinates)){ 
      return hashEntry.pointer;
    }
  }
}

__device__
float getMagnitude(Vector3f vector){
  return sqrt(pow(vector(0),2) + pow(vector(1),2) + pow(vector(2),2));
}

__device__
float dotProduct(Vector3f a, Vector3f b){
  return a(0)*b(0) + a(1)*b(1) + a(2)*b(2);
}


__device__
float getDistanceUpdate(Vector3f voxelCoordinates, Vector3f point_d, Vector3f origin){
//x:center of current voxel   p:position of lidar point   s:sensor origin
Vector3f lidarOriginDiff = point_d - origin; //p-s
Vector3f lidarVoxelDiff = point_d - voxelCoordinates; //p-x
float magnitudeLidarVoxelDiff = getMagnitude(lidarVoxelDiff);
float dotProd = dotProduct(lidarOriginDiff, lidarVoxelDiff);
if(dotProd < 0){
  magnitudeLidarVoxelDiff *=-1;
}

return magnitudeLidarVoxelDiff;
}

__global__
void updateVoxels(Vector3f * voxels, HashTable * hashTable_d, BlockHeap * blockHeap_d, Vector3f * point_d, Vector3f * origin){
  int threadIndex = threadIdx.x;
  Vector3f voxelCoordinates = voxels[threadIndex];
  Vector3f voxelBlockCoordinates = GetVoxelBlockCenterFromPoint(voxelCoordinates);
  //make a global vector with half voxel block size values
  int voxelBlockHeapPosition = getBlockPositionForBlockCoordinates(voxelBlockCoordinates, hashTable_d);
  Vector3f voxelBlockBottomLeftCoordinates;
  voxelBlockBottomLeftCoordinates(0) = voxelBlockCoordinates(0)-HALF_VOXEL_BLOCK_SIZE;
  voxelBlockBottomLeftCoordinates(1) = voxelBlockCoordinates(1)-HALF_VOXEL_BLOCK_SIZE;
  voxelBlockBottomLeftCoordinates(2) = voxelBlockCoordinates(2)-HALF_VOXEL_BLOCK_SIZE;
  size_t localVoxelIndex = getLocalVoxelIndex(voxelCoordinates - voxelBlockBottomLeftCoordinates);
  Vector3f point = * point_d;
  VoxelBlock * block = &(blockHeap_d->blocks[voxelBlockHeapPosition]);
  Voxel* voxel = &(block->voxels[localVoxelIndex]);
  int * mutex = &(block->mutex[localVoxelIndex]);

  float weight = 1;
  float distance = getDistanceUpdate(voxelCoordinates, *point_d, *origin);
  float weightTimesDistance = weight * distance;

  //get lock for voxel
  bool updatedVoxel = false;
  while(!updatedVoxel){
    if(!atomicCAS(mutex, 0, 1)){
      updatedVoxel = true;
      //update sdf and weight
      float oldWeight = voxel->weight;
      float oldSdf = voxel->sdf;
      float newWeight = oldWeight + weight;
      float newDistance = (oldWeight * oldSdf + weightTimesDistance) / newWeight;
      voxel->sdf = newDistance;
      newWeight = min(newWeight, MAX_WEIGHT);
      voxel->weight = newWeight;
      // printf("voxel coords: (%f, %f, %f) with sdf: %f with weight: %f\n", voxelCoordinates(0),voxelCoordinates(1), voxelCoordinates(2), voxel->sdf, voxel->weight);
      atomicExch(mutex, 0);
    }
  }
}

__global__
void getVoxelsForPoint(pcl::PointXYZ * points_d, Vector3f * origin_transformed_d, HashTable * hashTable_d, BlockHeap * blockHeap_d, int * size_d){
 int threadIndex = (blockIdx.x*128 + threadIdx.x);
 if(threadIndex>=*size_d){
  return;
}
 pcl::PointXYZ point_d = points_d[threadIndex];
 Vector3f u = *origin_transformed_d;
 Vector3f point_d_vector(point_d.x, point_d.y, point_d.z);
 Vector3f v = point_d_vector - u; //direction
 //equation of line is u+tv
 float vMag = sqrt(pow(v(0), 2) + pow(v(1),2) + pow(v(2), 2));
 Vector3f v_normalized = v / vMag;
 Vector3f truncation_start = point_d_vector - truncation_distance*v_normalized;
 
 Vector3f truncation_end = point_d_vector + truncation_distance*v_normalized;

 float distance_tStart_origin = sqrt(pow(truncation_start(0) - u(0), 2) + pow(truncation_start(1) - u(1),2) + pow(truncation_start(2) - u(2), 2));
 float distance_tEnd_origin = sqrt(pow(truncation_end(0) - u(0), 2) + pow(truncation_end(1) - u(1),2) + pow(truncation_end(2) - u(2), 2));

 if(distance_tEnd_origin < distance_tStart_origin){
   Vector3f temp = truncation_start;
   truncation_start = truncation_end;
   truncation_end = temp;
 }

 Vector3f truncation_start_voxel = GetVoxelCenterFromPoint(truncation_start);
 Vector3f truncation_end_voxel = GetVoxelCenterFromPoint(truncation_end);
 float stepX = v(0) > 0 ? VOXEL_SIZE : -1 * VOXEL_SIZE;
 float stepY = v(1) > 0 ? VOXEL_SIZE : -1 * VOXEL_SIZE;
 float stepZ = v(2) > 0 ? VOXEL_SIZE : -1 * VOXEL_SIZE;
 float tMaxX = fabs(v(0) < 0 ? (truncation_start_voxel(0) - HALF_VOXEL_SIZE - u(0)) / v(0) : (truncation_start_voxel(0) + HALF_VOXEL_SIZE - u(0)) / v(0));
 float tMaxY = fabs(v(1) < 0 ? (truncation_start_voxel(1) - HALF_VOXEL_SIZE - u(1)) / v(1) : (truncation_start_voxel(1) + HALF_VOXEL_SIZE - u(1)) / v(1));
 float tMaxZ = fabs(v(2) < 0 ? (truncation_start_voxel(2) - HALF_VOXEL_SIZE - u(2)) / v(2) : (truncation_start_voxel(2) + HALF_VOXEL_SIZE - u(2)) / v(2));
 float tDeltaX = fabs(VOXEL_SIZE / v(0));
 float tDeltaY = fabs(VOXEL_SIZE / v(1));
 float tDeltaZ = fabs(VOXEL_SIZE / v(2));
 Vector3f currentBlock(truncation_start_voxel(0), truncation_start_voxel(1), truncation_start_voxel(2));

 //overkill - how big should this be?
 Vector3f * voxels = new Vector3f[200]; //set in terms of truncation distance and voxel size
 int size = 0;
 while(!checkFloatingPointVectorsEqual(currentBlock, truncation_end_voxel, VOXEL_EPSILON)){
   voxels[size] = currentBlock;
   size++;
  if(tMaxX < tMaxY){
    if(tMaxX < tMaxZ)
    {
      currentBlock(0) += stepX;
      tMaxX += tDeltaX;
    }
    else if(tMaxX > tMaxZ){
      currentBlock(2) += stepZ;
      tMaxZ += tDeltaZ;
    }
    else{
      currentBlock(0) += stepX;
      currentBlock(2) += stepZ;
      tMaxX += tDeltaX;
      tMaxZ += tDeltaZ;
    }
  }
  else if(tMaxX > tMaxY){
    if(tMaxY < tMaxZ){
      currentBlock(1) += stepY;
      tMaxY += tDeltaY;
    }
    else if(tMaxY > tMaxZ){
      currentBlock(2) += stepZ;
      tMaxZ += tDeltaZ;
    }
    else{
      currentBlock(1) += stepY;
      currentBlock(2) += stepZ;
      tMaxY += tDeltaY;
      tMaxZ += tDeltaZ;
    }
  }
  else{
    if(tMaxZ < tMaxX){
      currentBlock(2) += stepZ;
      tMaxZ += tDeltaZ;
    }
    else if(tMaxZ > tMaxX){
      currentBlock(0) += stepX;
      currentBlock(1) += stepY;
      tMaxX += tDeltaX;
      tMaxY += tDeltaY;
    }
    else{
      currentBlock(0) += stepX;
      currentBlock(1) += stepY;
      currentBlock(2) += stepZ;
      tMaxX += tDeltaX;
      tMaxY += tDeltaY;
      tMaxZ += tDeltaZ;
    }
  }   
 }   

 voxels[size] = currentBlock;
 size++;

 Vector3f * lidarPoint = new Vector3f(point_d.x, point_d.y, point_d.z);
 //update to check if size is greater than threads per block
 updateVoxels<<<1, size>>>(voxels, hashTable_d, blockHeap_d, lidarPoint, origin_transformed_d);
 cdpErrchk(hipPeekAtLastError());
 hipDeviceSynchronize();
 hipFree(lidarPoint);
  hipFree(voxels);
  return;
}

__global__
void processOccupiedVoxelBlock(Vector3f * occupiedVoxels, int * index, Vector3f * position, VoxelBlock * block){
  int threadIndex = blockIdx.x*128 + threadIdx.x;
  if(threadIndex >= VOXEL_PER_BLOCK * VOXEL_PER_BLOCK * VOXEL_PER_BLOCK){
    return;
  }

  int voxelIndex = threadIndex;
  Voxel voxel = block->voxels[threadIndex];
  if(voxel.weight!=0){
    float z = voxelIndex / (VOXEL_PER_BLOCK * VOXEL_PER_BLOCK);
    voxelIndex -= z*VOXEL_PER_BLOCK*VOXEL_PER_BLOCK;
    float y = voxelIndex / VOXEL_PER_BLOCK;
    voxelIndex -= y*VOXEL_PER_BLOCK;
    float x = voxelIndex;

    Vector3f positionVec = * position;
    float xCoord = x * VOXEL_SIZE + HALF_VOXEL_SIZE + positionVec(0);
    float yCoord = y * VOXEL_SIZE + HALF_VOXEL_SIZE + positionVec(1);
    float zCoord = z * VOXEL_SIZE + HALF_VOXEL_SIZE + positionVec(2);
  
    Vector3f v(xCoord, yCoord, zCoord);
    int occupiedVoxelIndex = atomicAdd(&(*index), 1);
    occupiedVoxels[occupiedVoxelIndex] = v;
  }
}

__global__
void visualizeOccupiedVoxels(HashTable * hashTable_d, BlockHeap * blockHeap_d, Vector3f * occupiedVoxels, int * index){
  int threadIndex = blockIdx.x*128 +threadIdx.x;
  if(threadIndex >= HASH_TABLE_SIZE) return;
  HashEntry hashEntry = hashTable_d->hashEntries[threadIndex];
  if(hashEntry.isFree()){
    return;
  }
  int pointer = hashEntry.pointer;
  Vector3f * position = new Vector3f(hashEntry.position(0) - HALF_VOXEL_BLOCK_SIZE, 
  hashEntry.position(1)- HALF_VOXEL_BLOCK_SIZE,
  hashEntry.position(2)- HALF_VOXEL_BLOCK_SIZE);

  VoxelBlock * block = &(blockHeap_d->blocks[pointer]);
  int size = VOXEL_PER_BLOCK * VOXEL_PER_BLOCK * VOXEL_PER_BLOCK;
  int numBlocks = size/128 + 1;
  processOccupiedVoxelBlock<<<numBlocks,128>>>(occupiedVoxels, index, position, block);
  cdpErrchk(hipPeekAtLastError());
  hipFree(position);
}

TSDFHandler::TSDFHandler(){
  tsdfContainer = new TSDFContainer();
}

TSDFHandler::~TSDFHandler(){
  free(tsdfContainer);
}

void TSDFHandler::integrateVoxelBlockPointsIntoHashTable(Vector3f points_h[], int size, HashTable * hashTable_d, BlockHeap * blockHeap_d){
  int * size_h = &size;
  int * size_d;
  hipMalloc(&size_d, sizeof(*size_h));
  hipMemcpy(size_d, size_h, sizeof(*size_h), hipMemcpyHostToDevice);

  bool * unallocatedPoints_h = new bool[size];
  for(int i=0;i<size;++i)
  {
    unallocatedPoints_h[i] = 1;
  }
  bool * unallocatedPoints_d;
  hipMalloc(&unallocatedPoints_d, sizeof(*unallocatedPoints_h)*size);
  hipMemcpy(unallocatedPoints_d, unallocatedPoints_h, sizeof(*unallocatedPoints_h)*size, hipMemcpyHostToDevice);

  int * unallocatedPointsCount_h = new int(size);
  int * unallocatedPointsCount_d;
  hipMalloc(&unallocatedPointsCount_d, sizeof(*unallocatedPointsCount_h));
  hipMemcpy(unallocatedPointsCount_d, unallocatedPointsCount_h, sizeof(*unallocatedPointsCount_h), hipMemcpyHostToDevice);

  Vector3f * points_d;
  hipMalloc(&points_d, sizeof(*points_h)*size);
  hipMemcpy(points_d, points_h, sizeof(*points_h)*size, hipMemcpyHostToDevice);

  int numCudaBlocks = size / threadsPerCudaBlock + 1;
  while(*unallocatedPointsCount_h > 0){ //FIX THIS SO THERE IS NO POSSIBILITY OF INFINITE LOOP WHEN INSERTING INTO THE HASH TABLE IS NOT POSSIBLE - check size of block heap pointer or whether hash table is full in available entries for inserting a point
    allocateVoxelBlocks<<<numCudaBlocks,threadsPerCudaBlock>>>(points_d, hashTable_d, blockHeap_d, unallocatedPoints_d, size_d, unallocatedPointsCount_d);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();
    hipMemcpy(unallocatedPointsCount_h, unallocatedPointsCount_d, sizeof(*unallocatedPointsCount_h), hipMemcpyDeviceToHost);
  }

  printHashTableAndBlockHeap<<<1,1>>>(hashTable_d, blockHeap_d);
  hipDeviceSynchronize();

  hipFree(size_d);
  hipFree(unallocatedPoints_d);
  hipFree(unallocatedPointsCount_d);
  hipFree(points_d);
  free(unallocatedPoints_h);
  free(unallocatedPointsCount_h);
}

void TSDFHandler::processPointCloudAndUpdateVoxels(pcl::PointCloud<pcl::PointXYZ>::Ptr pointcloud, Vector3f * origin_transformed_h, Vector3f * occupiedVoxels_h, int * index_h)
{
  
  std::vector<pcl::PointXYZ, Eigen::aligned_allocator<pcl::PointXYZ>> points = pointcloud->points;

  pcl::PointXYZ * points_h = &points[0];
  pcl::PointXYZ * points_d;
  int size = pointcloud->size();
  int * size_d;
  hipMalloc(&size_d, sizeof(int));
  hipMemcpy(size_d, &size, sizeof(int), hipMemcpyHostToDevice);

  hipMalloc(&points_d, sizeof(*points_h)*size);
  hipMemcpy(points_d, points_h, sizeof(*points_h)*size, hipMemcpyHostToDevice);

  //TODO: FIX
  // int maxBlocksPerPoint = ceil(pow(truncation_distance,3) / pow(VOXEL_BLOCK_SIZE, 3));
  int maxBlocks = 10 * size;
  Vector3f pointCloudVoxelBlocks_h[maxBlocks];
  Vector3f * pointCloudVoxelBlocks_d;
  int * pointer_h = new int(0);
  int * pointer_d;
  hipMalloc(&pointCloudVoxelBlocks_d, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks);
  hipMemcpy(pointCloudVoxelBlocks_d, pointCloudVoxelBlocks_h, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks,hipMemcpyHostToDevice); //do I even need to memcpy
  hipMalloc(&pointer_d, sizeof(*pointer_h));
  hipMemcpy(pointer_d, pointer_h, sizeof(*pointer_h), hipMemcpyHostToDevice);

  Vector3f * origin_transformed_d;
  hipMalloc(&origin_transformed_d, sizeof(*origin_transformed_h));
  hipMemcpy(origin_transformed_d, origin_transformed_h,sizeof(*origin_transformed_h),hipMemcpyHostToDevice);

  int numCudaBlocks = size / threadsPerCudaBlock + 1;
  //since size can go over threads per block allocate this properly to include all data
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  getVoxelBlocksForPoint<<<numCudaBlocks,threadsPerCudaBlock>>>(points_d, pointCloudVoxelBlocks_d, pointer_d, origin_transformed_d, size_d);
  gpuErrchk( hipPeekAtLastError() );
  hipDeviceSynchronize();
  //NOT NECESSARY - CHANGE THIS !
  hipMemcpy(pointCloudVoxelBlocks_h, pointCloudVoxelBlocks_d, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks,hipMemcpyDeviceToHost);
  hipMemcpy(pointer_h, pointer_d, sizeof(*pointer_h), hipMemcpyDeviceToHost);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("get voxel block duration: %f\n", milliseconds);

  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  hipEventRecord(start1);

  HashTable * hashTable_d = tsdfContainer->getCudaHashTable();

  BlockHeap * blockHeap_d = tsdfContainer->getCudaBlockHeap();

  integrateVoxelBlockPointsIntoHashTable(pointCloudVoxelBlocks_h, *pointer_h, hashTable_d, blockHeap_d);

  hipEventRecord(stop1);
  hipEventSynchronize(stop1);
  float milliseconds1 = 0;
  hipEventElapsedTime(&milliseconds1, start1, stop1);
  printf("integrate voxel block duration: %f\n", milliseconds1);
  hipEvent_t start2, stop2;
  hipEventCreate(&start2);
  hipEventCreate(&stop2);
  hipEventRecord(start2);
  getVoxelsForPoint<<<numCudaBlocks,threadsPerCudaBlock>>>(points_d, origin_transformed_d, hashTable_d, blockHeap_d, size_d);
  gpuErrchk( hipPeekAtLastError() );
  hipDeviceSynchronize();

  Vector3f * occupiedVoxels_d;
  int * index_d;
  int occupiedVoxelsSize = 1000000;
  hipMalloc(&occupiedVoxels_d, sizeof(*occupiedVoxels_h)*occupiedVoxelsSize);
  hipMemcpy(occupiedVoxels_d, occupiedVoxels_h, sizeof(*occupiedVoxels_h)*occupiedVoxelsSize,hipMemcpyHostToDevice);
  hipMalloc(&index_d, sizeof(*index_h));
  hipMemcpy(index_d, index_h, sizeof(*index_h), hipMemcpyHostToDevice);

  int numVisVoxBlocks = HASH_TABLE_SIZE / threadsPerCudaBlock + 1;
  // printf("hash table size: %d\n", HASH_TABLE_SIZE);
  visualizeOccupiedVoxels<<<numVisVoxBlocks,threadsPerCudaBlock>>>(hashTable_d, blockHeap_d, occupiedVoxels_d, index_d);
  gpuErrchk( hipPeekAtLastError() );
  hipDeviceSynchronize();

  hipMemcpy(occupiedVoxels_h, occupiedVoxels_d, sizeof(*occupiedVoxels_h)*occupiedVoxelsSize, hipMemcpyDeviceToHost);
  hipMemcpy(index_h, index_d, sizeof(int), hipMemcpyDeviceToHost);

  printf("size of occupied voxels: %d\n", *index_h);

  hipFree(size_d);
  hipFree(points_d);
  hipFree(pointCloudVoxelBlocks_d);
  hipFree(pointer_d);
  hipFree(origin_transformed_d);
  hipFree(occupiedVoxels_d); //instead of allocating and freeing over and over just add to tsdfhandler
  hipFree(index_d);

  hipEventRecord(stop2);
  hipEventSynchronize(stop2);
  float milliseconds2 = 0;
  hipEventElapsedTime(&milliseconds2, start2, stop2);
  printf("update voxels duration: %f\n", milliseconds2);

  free(pointer_h);

}

//takes sensor origin position
void pointcloudMain(pcl::PointCloud<pcl::PointXYZ>::Ptr pointcloud, Vector3f * origin_transformed_h, TSDFContainer * tsdfContainer, Vector3f * occupiedVoxels_h, int * index_h)
{
  //retrieve sensor origin..can use transformation from point cloud time stamp drone_1/lidar frame to drone_1 frame then transform 0,0,0
  
  std::vector<pcl::PointXYZ, Eigen::aligned_allocator<pcl::PointXYZ>> points = pointcloud->points;

  pcl::PointXYZ * points_h = &points[0];
  pcl::PointXYZ * points_d;
  int size = pointcloud->size();
  //printf("point c size: %d\n", size); //this works
  int * size_d;
  hipMalloc(&size_d, sizeof(int));
  hipMemcpy(size_d, &size, sizeof(int), hipMemcpyHostToDevice);

  hipMalloc(&points_d, sizeof(*points_h)*size);
  hipMemcpy(points_d, points_h, sizeof(*points_h)*size, hipMemcpyHostToDevice);

  // int * pointCloudVoxelBlockSize_h = new int(maxBlocksPerPoint * size);
  // int * pointCloudVoxelBlockSize_d;
  // hipMalloc(&pointCloudVoxelBlockSize_d, sizeof(int));
  // hipMemcpy(pointCloudVoxelBlockSize_d, pointCloudVoxelBlockSize_h, sizeof(int, ))
  //TODO: FIX
  // int maxBlocksPerPoint = ceil(pow(truncation_distance,3) / pow(VOXEL_BLOCK_SIZE, 3));
  int maxBlocks = 10 * size;
  // printf("maxBlocks: %d", maxBlocks);
  Vector3f pointCloudVoxelBlocks_h[maxBlocks];
  Vector3f * pointCloudVoxelBlocks_d;
  int * pointer_h = new int(0);
  int * pointer_d;
  hipMalloc(&pointCloudVoxelBlocks_d, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks);
  hipMemcpy(pointCloudVoxelBlocks_d, pointCloudVoxelBlocks_h, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks,hipMemcpyHostToDevice); //do I even need to memcpy
  hipMalloc(&pointer_d, sizeof(*pointer_h));
  hipMemcpy(pointer_d, pointer_h, sizeof(*pointer_h), hipMemcpyHostToDevice);

  Vector3f * origin_transformed_d;
  hipMalloc(&origin_transformed_d, sizeof(*origin_transformed_h));
  hipMemcpy(origin_transformed_d, origin_transformed_h,sizeof(*origin_transformed_h),hipMemcpyHostToDevice);
  // PointCloudVoxelBlocks * pointCloudVoxelBlocks_h = new PointCloudVoxelBlocks(maxBlocks);
  // PointCloudVoxelBlocks * pointCloudVoxelBlocks_d;

  // hipMalloc(&pointCloudVoxelBlocks_d, sizeof(float)*3*maxBlocks+4);
  // hipMemcpy(pointCloudVoxelBlocks_d, pointCloudVoxelBlocks_h, sizeof(Vector3f)*maxBlocks+sizeof(int), hipMemcpyHostToDevice);

  int numCudaBlocks = size / threadsPerCudaBlock + 1;
  //since size can go over threads per block allocate this properly to include all data
  // auto start1 = std::chrono::high_resolution_clock::now();
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  getVoxelBlocksForPoint<<<numCudaBlocks,threadsPerCudaBlock>>>(points_d, pointCloudVoxelBlocks_d, pointer_d, origin_transformed_d, size_d);
  gpuErrchk( hipPeekAtLastError() );
  hipDeviceSynchronize();
  // auto stop1 = std::chrono::high_resolution_clock::now();
  // auto duration1 = std::chrono::duration_cast<std::chrono::milliseconds>(stop1 - start1); 
  // std::cout << "get voxel blocks duration: ";
  // std::cout << duration1.count() << std::endl; 
  //NOT NECESSARY - CHANGE THIS !
  hipMemcpy(pointCloudVoxelBlocks_h, pointCloudVoxelBlocks_d, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks,hipMemcpyDeviceToHost);
  hipMemcpy(pointer_h, pointer_d, sizeof(*pointer_h), hipMemcpyDeviceToHost);

  // printf("point: (%f,%f,%f)\n", pointCloudVoxelBlocks_h[0](0), pointCloudVoxelBlocks_h[0](1), pointCloudVoxelBlocks_h[0](2));
  // printf("int val: %d\n", *pointer_h);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("get voxel block duration: %f\n", milliseconds);


  // printf("size: %d\n", *pointer_h);
  // auto start2 = std::chrono::high_resolution_clock::now();
  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  hipEventRecord(start1);
  // tsdfContainer->integrateVoxelBlockPointsIntoHashTable(pointCloudVoxelBlocks_h, *pointer_h);
  // auto stop2 = std::chrono::high_resolution_clock::now();
  // auto duration2 = std::chrono::duration_cast<std::chrono::milliseconds>(stop2 - start2); 
  // std::cout<< "integrate voxel blocks duration: ";
  // std::cout << duration2.count() << std::endl; 
  // hipDeviceSynchronize();

  HashTable * hashTable_d = tsdfContainer->getCudaHashTable();

  BlockHeap * blockHeap_d = tsdfContainer->getCudaBlockHeap();

  hipEventRecord(stop1);
  hipEventSynchronize(stop1);
  float milliseconds1 = 0;
  hipEventElapsedTime(&milliseconds1, start1, stop1);
  printf("integrate voxel block duration: %f\n", milliseconds1);
  // auto start3 = std::chrono::high_resolution_clock::now();
  hipEvent_t start2, stop2;
  hipEventCreate(&start2);
  hipEventCreate(&stop2);
  hipEventRecord(start2);
  getVoxelsForPoint<<<numCudaBlocks,threadsPerCudaBlock>>>(points_d, origin_transformed_d, hashTable_d, blockHeap_d, size_d);
  gpuErrchk( hipPeekAtLastError() );
  hipDeviceSynchronize();
  // auto stop3 = std::chrono::high_resolution_clock::now();
  // auto duration3 = std::chrono::duration_cast<std::chrono::milliseconds>(stop3 - start3); 
  // std::cout<< "update voxels duration: ";
  // std::cout << duration3.count() << std::endl; 

  Vector3f * occupiedVoxels_d;
  int * index_d;
  int occupiedVoxelsSize = 1000000;
  hipMalloc(&occupiedVoxels_d, sizeof(*occupiedVoxels_h)*occupiedVoxelsSize);
  hipMemcpy(occupiedVoxels_d, occupiedVoxels_h, sizeof(*occupiedVoxels_h)*occupiedVoxelsSize,hipMemcpyHostToDevice);
  hipMalloc(&index_d, sizeof(*index_h));
  hipMemcpy(index_d, index_h, sizeof(*index_h), hipMemcpyHostToDevice);

  int numVisVoxBlocks = HASH_TABLE_SIZE / threadsPerCudaBlock + 1;
  // printf("hash table size: %d\n", HASH_TABLE_SIZE);
  visualizeOccupiedVoxels<<<numVisVoxBlocks,threadsPerCudaBlock>>>(hashTable_d, blockHeap_d, occupiedVoxels_d, index_d);
  gpuErrchk( hipPeekAtLastError() );
  hipDeviceSynchronize();

  hipMemcpy(occupiedVoxels_h, occupiedVoxels_d, sizeof(*occupiedVoxels_h)*occupiedVoxelsSize, hipMemcpyDeviceToHost);
  hipMemcpy(index_h, index_d, sizeof(int), hipMemcpyDeviceToHost);

  printf("size of occupied voxels: %d\n", *index_h);

  hipFree(size_d);
  hipFree(points_d);
  hipFree(pointCloudVoxelBlocks_d);
  hipFree(pointer_d);
  hipFree(origin_transformed_d);
  hipFree(occupiedVoxels_d); //instead of allocating and freeing over and over just add to tsdfhandler
  hipFree(index_d);

  hipEventRecord(stop2);
  hipEventSynchronize(stop2);
  float milliseconds2 = 0;
  hipEventElapsedTime(&milliseconds2, start2, stop2);
  printf("update voxels duration: %f\n", milliseconds2);

  free(pointer_h);

  // for(size_t i=0; i<points.size(); ++i){
  //     printf("Cloud with Points: %f, %f, %f\n", points[i].x,points[i].y,points[i].z);
  //   } 
}

void testVoxelBlockTraversal(TSDFContainer * tsdfContainer, Vector3f * occupiedVoxels_h, int * index_h){
  // float f = 10.23423;
  int size = 5;
  pcl::PointXYZ * point1 = new pcl::PointXYZ(.75,.75, 0.75);
  pcl::PointXYZ * point2 = new pcl::PointXYZ(1.5,1.5, 1.5);
  pcl::PointXYZ * point3 = new pcl::PointXYZ(-.85,-.85, -.85);
  pcl::PointXYZ * point4 = new pcl::PointXYZ(.90,.90, .90);
  pcl::PointXYZ * point5 = new pcl::PointXYZ(.65,.56, .65);
  
  pcl::PointXYZ * points_h = new pcl::PointXYZ[size];
  points_h[0] = *point1;
  points_h[1] = *point2;
  points_h[2] = *point3;
  points_h[3] = *point4;
  points_h[4] = *point5;
  pcl::PointXYZ * points_d;
  hipMalloc(&points_d, sizeof(*points_h)*size);
  hipMemcpy(points_d, points_h, sizeof(*points_h)*size, hipMemcpyHostToDevice);

  int maxBlocks = 1000;
  Vector3f pointCloudVoxelBlocks_h[maxBlocks]; //make these member functions of tsdf_handler if cant pass device reference on host code
  Vector3f * pointCloudVoxelBlocks_d;
  int * pointer_h = new int(0);
  int * pointer_d;
  hipMalloc(&pointCloudVoxelBlocks_d, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks);
  hipMemcpy(pointCloudVoxelBlocks_d, pointCloudVoxelBlocks_h, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks,hipMemcpyHostToDevice); //do I even need to memcpy
  hipMalloc(&pointer_d, sizeof(*pointer_h));
  hipMemcpy(pointer_d, pointer_h, sizeof(*pointer_h), hipMemcpyHostToDevice);

  Vector3f * origin_transformed_h = new Vector3f(0,0,0);
  Vector3f * origin_transformed_d;
  hipMalloc(&origin_transformed_d, sizeof(*origin_transformed_h));
  hipMemcpy(origin_transformed_d, origin_transformed_h,sizeof(*origin_transformed_h),hipMemcpyHostToDevice);

  int numCudaBlocks = size / threadsPerCudaBlock + 1;

  int * size_d;
  hipMalloc(&size_d, sizeof(int));
  hipMemcpy(size_d, &size, sizeof(int), hipMemcpyHostToDevice);

  getVoxelBlocksForPoint<<<numCudaBlocks,threadsPerCudaBlock>>>(points_d, pointCloudVoxelBlocks_d, pointer_d, origin_transformed_d, size_d);

  // printVoxelBlocksFromPoint<<<1,1>>>(pointCloudVoxelBlocks_d, pointer_d);

  hipDeviceSynchronize();

  hipMemcpy(pointCloudVoxelBlocks_h, pointCloudVoxelBlocks_d, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks,hipMemcpyDeviceToHost);
  hipMemcpy(pointer_h, pointer_d, sizeof(*pointer_h), hipMemcpyDeviceToHost);

  printf("num voxel blocks: %d\n", *pointer_h);

  // tsdfContainer->integrateVoxelBlockPointsIntoHashTable(pointCloudVoxelBlocks_h, *pointer_h);

  HashTable * hashTable_d = tsdfContainer->getCudaHashTable();

  BlockHeap * blockHeap_d = tsdfContainer->getCudaBlockHeap();
  getVoxelsForPoint<<<numCudaBlocks,threadsPerCudaBlock>>>(points_d, origin_transformed_d, hashTable_d, blockHeap_d, size_d);

  hipDeviceSynchronize();

  Vector3f * occupiedVoxels_d;
  int * index_d;
  hipMalloc(&occupiedVoxels_d, sizeof(*occupiedVoxels_h)*100);
  hipMemcpy(occupiedVoxels_d, occupiedVoxels_h, sizeof(*occupiedVoxels_h)*100,hipMemcpyHostToDevice);
  hipMalloc(&index_d, sizeof(*index_h));
  hipMemcpy(index_d, index_h, sizeof(*index_h), hipMemcpyHostToDevice);

  int numVisVoxBlocks = HASH_TABLE_SIZE / 128 + 1;
  // printf("hash table size: %d\n", HASH_TABLE_SIZE);
  visualizeOccupiedVoxels<<<numVisVoxBlocks,128>>>(hashTable_d, blockHeap_d, occupiedVoxels_d, index_d);

  hipDeviceSynchronize();

  hipMemcpy(occupiedVoxels_h, occupiedVoxels_d, sizeof(*occupiedVoxels_h)*100, hipMemcpyDeviceToHost);
  hipMemcpy(index_h, index_d, sizeof(int), hipMemcpyDeviceToHost);

  // for(int i=0; i < *index_h; ++i){
  //   printf("occupied voxel: (%f, %f, %f)\n", occupiedVoxels_h[i](0), occupiedVoxels_h[i](1), occupiedVoxels_h[i](2));
  // }

  // printf("occupied voxels: %d\n", *index_h);

}

void testVoxelTraversal(){
  int size = 2;
  pcl::PointXYZ * point1 = new pcl::PointXYZ(-73.4567,33.576, 632.8910);
  pcl::PointXYZ * point2 = new pcl::PointXYZ(-7.23421,-278, 576.2342);
  pcl::PointXYZ * points_h = new pcl::PointXYZ[size];
  points_h[0] = *point1;
  points_h[1] = *point2;
  pcl::PointXYZ * points_d;
  hipMalloc(&points_d, sizeof(*points_h)*size);
  hipMemcpy(points_d, points_h, sizeof(*points_h)*size, hipMemcpyHostToDevice);

  

  // Vector3f * voxels_h = new Vector3f[size];
  // Vector3f * voxels_d;
  // Vector3f A;
  // A(0) = -1*(VOXEL_SIZE/2);
  // A(1) = -1*(VOXEL_SIZE/2);
  // A(2) = -1*(VOXEL_SIZE/2);
  // voxels_h[0] = A;
  // // Vector3f B;
  // // B(0) = 10/26;
  // // B(1) = 5/26;
  // // B(2) = 5/26;
  // // voxels_h[1] = B;
  // // Vector3f C;
  // // C(0) = 5/26;
  // // C(1) = 10/26;
  // // C(2) = 5/26;
  // // voxels_h[2] = C;
  // // Vector3f D;
  // // D(0) = -0.75;
  // // D(1) = -0.75;
  // // D(2) = -0.25;
  // // voxels_h[3] = D;
  // // Vector3f E;
  // // E(0) = -0.25;
  // // E(1) = -0.25;
  // // E(2) = -0.75;
  // // voxels_h[4] = E;
  // // Vector3f F;
  // // F(0) = -0.75;
  // // F(1) = -0.25;
  // // F(2) = -0.75;
  // // voxels_h[5] = F;
  // // Vector3f G;
  // // G(0) = -0.25;
  // // G(1) = -0.75;
  // // G(2) = -0.75;
  // // voxels_h[6] = G;
  // // Vector3f H;
  // // H(0) = -0.05;
  // // H(1) = -0.05;
  // // H(2) = -0.05;
  // // voxels_h[7] = H;
  

  // hipMalloc(&voxels_d, sizeof(*voxels_h)*size);
  // hipMemcpy(voxels_d,voxels_h, sizeof(*voxels_h)*size, hipMemcpyHostToDevice);

  // updateVoxels<<<1,size>>>(voxels_d);

  // hipDeviceSynchronize();

}