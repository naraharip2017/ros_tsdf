#include "hip/hip_runtime.h"
#include "cuda/tsdf_handler.cuh"

const int threadsPerCudaBlock = 128;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define cdpErrchk(ans) { cdpAssert((ans), __FILE__, __LINE__); }
__device__ void cdpAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      printf("GPU kernel assert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) assert(0);
   }
}

__global__
void printHashTableAndBlockHeap(HashTable * hashTable_d, BlockHeap * blockHeap_d){
  // HashEntry * hashEntries = hashTable_d->hashEntries;
  // for(size_t i=0;i<NUM_BUCKETS; ++i){
  //   printf("Bucket: %lu\n", (unsigned long)i);
  //   for(size_t it = 0; it<HASH_ENTRIES_PER_BUCKET; ++it){
  //     HashEntry hashEntry = hashEntries[it+i*HASH_ENTRIES_PER_BUCKET];
  //     Vector3f position = hashEntry.position;
  //     if (hashEntry.isFree()){
  //       printf("  Hash Entry with   Position: (N,N,N)   Offset: %d   Pointer: %d\n", hashEntry.offset, hashEntry.pointer);
  //     }
  //     else{
  //       printf("  Hash Entry with   Position: (%f,%f,%f)   Offset: %d   Pointer: %d\n", position(0), position(1), position(2), hashEntry.offset, hashEntry.pointer);
  //     }
  //   }
  //   printf("%s\n", "--------------------------------------------------------");
  // }

  // printf("Block Heap Free List: ");
  // int * freeBlocks = blockHeap_d->freeBlocks;
  // for(size_t i = 0; i<NUM_HEAP_BLOCKS; ++i){
  //   printf("%d  ", freeBlocks[i]);
  // }
  // printf("\n");
  printf("Current Index: %d\n", blockHeap_d->currentIndex);
}

__device__
size_t retrieveHashIndexFromPoint(Vector3f point){ //tested using int can get negatives
  return abs((((int)point(0)*PRIME_ONE) ^ ((int)point(1)*PRIME_TWO) ^ ((int)point(2)*PRIME_THREE)) % NUM_BUCKETS);
}

__device__ 
float FloorFun(float x, float scale){
  return floor(x*scale) / scale;
}

__device__
Vector3f GetVoxelBlockCenterFromPoint(Vector3f point){
  float scale = 1 / VOXEL_BLOCK_SIZE;
  Vector3f blockCenter;
  blockCenter(0) = FloorFun(point(0), scale) + HALF_VOXEL_BLOCK_SIZE;
  blockCenter(1) = FloorFun(point(1), scale) + HALF_VOXEL_BLOCK_SIZE;
  blockCenter(2) = FloorFun(point(2), scale) + HALF_VOXEL_BLOCK_SIZE;
  return blockCenter;
}

__device__
Vector3f GetVoxelCenterFromPoint(Vector3f point){
  float scale = 1 / VOXEL_SIZE;
  Vector3f voxelCenter;
  voxelCenter(0) = FloorFun(point(0), scale) + HALF_VOXEL_SIZE;
  voxelCenter(1) = FloorFun(point(1), scale) + HALF_VOXEL_SIZE;
  voxelCenter(2) = FloorFun(point(2), scale) + HALF_VOXEL_SIZE;
  return voxelCenter;
}

__device__
bool checkFloatingPointVectorsEqual(Vector3f A, Vector3f B, float epsilon){
  Vector3f diff = A-B;
  //have to use an epsilon value due to floating point precision errors
  if((fabs(diff(0)) < epsilon) && (fabs(diff(1)) < epsilon) && (fabs(diff(2)) < epsilon))
    return true;

  return false;
}

__device__
inline void getTruncationLineEndPoints(pcl::PointXYZ & point_d, Vector3f * origin_transformed_d, Vector3f & truncation_start, Vector3f & truncation_end, Vector3f & u, Vector3f & v){
  u = *origin_transformed_d;
  Vector3f point_d_vector(point_d.x, point_d.y, point_d.z);
  v = point_d_vector - u; //direction
  //equation of line is u+tv
  float vMag = sqrt(pow(v(0), 2) + pow(v(1),2) + pow(v(2), 2));
  Vector3f v_normalized = v / vMag;
  truncation_start = point_d_vector - truncation_distance*v_normalized;
  
  truncation_end = point_d_vector + truncation_distance*v_normalized;

  float distance_tStart_origin = pow(truncation_start(0) - u(0), 2) + pow(truncation_start(1) - u(1),2) + pow(truncation_start(2) - u(2), 2);
  float distance_tEnd_origin = pow(truncation_end(0) - u(0), 2) + pow(truncation_end(1) - u(1),2) + pow(truncation_end(2) - u(2), 2);

  if(distance_tEnd_origin < distance_tStart_origin){
    Vector3f temp = truncation_start;
    truncation_start = truncation_end;
    truncation_end = temp;
  }
}

__device__
inline void traverseVolume(Vector3f & truncation_start_vol, Vector3f & truncation_end_vol, const float & volume_size, Vector3f & u, Vector3f & v, Vector3f * traversed_vols, int * traversed_vols_size){
  float half_volume_size = volume_size / 2;
  const float epsilon = volume_size / 4;
  float stepX = v(0) > 0 ? volume_size : -1 * volume_size;
  float stepY = v(1) > 0 ? volume_size : -1 * volume_size;
  float stepZ = v(2) > 0 ? volume_size : -1 * volume_size;
  float tMaxX = fabs(v(0) < 0 ? (truncation_start_vol(0) - half_volume_size - u(0)) / v(0) : (truncation_start_vol(0) + half_volume_size - u(0)) / v(0));
  float tMaxY = fabs(v(1) < 0 ? (truncation_start_vol(1) - half_volume_size - u(1)) / v(1) : (truncation_start_vol(1) + half_volume_size - u(1)) / v(1));
  float tMaxZ = fabs(v(2) < 0 ? (truncation_start_vol(2) - half_volume_size - u(2)) / v(2) : (truncation_start_vol(2) + half_volume_size - u(2)) / v(2));
  float tDeltaX = fabs(volume_size / v(0));
  float tDeltaY = fabs(volume_size / v(1));
  float tDeltaZ = fabs(volume_size / v(2));
  Vector3f current_vol(truncation_start_vol(0), truncation_start_vol(1), truncation_start_vol(2));

  int insert_index;
  while(!checkFloatingPointVectorsEqual(current_vol, truncation_end_vol, epsilon)){
    insert_index = atomicAdd(&(*traversed_vols_size), 1);
    traversed_vols[insert_index] = current_vol;
    if(tMaxX < tMaxY){
      if(tMaxX < tMaxZ)
      {
        current_vol(0) += stepX;
        tMaxX += tDeltaX;
      }
      else if(tMaxX > tMaxZ){
        current_vol(2) += stepZ;
        tMaxZ += tDeltaZ;
      }
      else{
        current_vol(0) += stepX;
        current_vol(2) += stepZ;
        tMaxX += tDeltaX;
        tMaxZ += tDeltaZ;
      }
    }
    else if(tMaxX > tMaxY){
      if(tMaxY < tMaxZ){
        current_vol(1) += stepY;
        tMaxY += tDeltaY;
      }
      else if(tMaxY > tMaxZ){
        current_vol(2) += stepZ;
        tMaxZ += tDeltaZ;
      }
      else{
        current_vol(1) += stepY;
        current_vol(2) += stepZ;
        tMaxY += tDeltaY;
        tMaxZ += tDeltaZ;
      }
    }
    else{
      if(tMaxZ < tMaxX){
        current_vol(2) += stepZ;
        tMaxZ += tDeltaZ;
      }
      else if(tMaxZ > tMaxX){
        current_vol(0) += stepX;
        current_vol(1) += stepY;
        tMaxX += tDeltaX;
        tMaxY += tDeltaY;
      }
      else{ 
        current_vol(0) += stepX;
        current_vol(1) += stepY;
        current_vol(2) += stepZ;
        tMaxX += tDeltaX;
        tMaxY += tDeltaY;
        tMaxZ += tDeltaZ;
      }
    } 
  }      

  insert_index = atomicAdd(&(*traversed_vols_size), 1);
  traversed_vols[insert_index] = current_vol;
}

__global__
void getVoxelBlocksForPoint(pcl::PointXYZ * points_d, Vector3f * pointCloudVoxelBlocks_d, int * pointer_d, Vector3f * origin_transformed_d, int * size_d){ //refactor
  int threadIndex = (blockIdx.x*threadsPerCudaBlock + threadIdx.x);
  if(threadIndex>=*size_d){
    return;
  }
  pcl::PointXYZ point_d = points_d[threadIndex];
  Vector3f truncation_start;
  Vector3f truncation_end;
  Vector3f u;
  Vector3f v;

  getTruncationLineEndPoints(point_d, origin_transformed_d, truncation_start, truncation_end, u, v);

  Vector3f truncation_start_block = GetVoxelBlockCenterFromPoint(truncation_start);
  Vector3f truncation_end_block = GetVoxelBlockCenterFromPoint(truncation_end);

  traverseVolume(truncation_start_block, truncation_end_block, VOXEL_BLOCK_SIZE, u, v, pointCloudVoxelBlocks_d, pointer_d);
  return;
}

__device__ 
int getBlockPositionForBlockCoordinates(Vector3f & voxelBlockCoordinates, size_t & bucket_index, size_t & currentGlobalIndex, HashEntry * hashEntries){

  HashEntry hashEntry;

  //check the hashed bucket for the block
  for(size_t i=0; i<HASH_ENTRIES_PER_BUCKET; ++i){
    hashEntry = hashEntries[currentGlobalIndex+i];
    if(hashEntry.checkIsPositionEqual(voxelBlockCoordinates)){
      return hashEntry.pointer;
    }
  }

  currentGlobalIndex+=HASH_ENTRIES_PER_BUCKET-1;

  //check the linked list if necessary
  while(hashEntry.offset!=0){
    short offset = hashEntry.offset;
    currentGlobalIndex+=offset;
    if(currentGlobalIndex>=HASH_TABLE_SIZE){
      currentGlobalIndex %= HASH_TABLE_SIZE;
    }
    hashEntry = hashEntries[currentGlobalIndex];
    if(hashEntry.checkIsPositionEqual(voxelBlockCoordinates)){ 
      return hashEntry.pointer;
    }
  }

  //block not allocated in hashTable
  return -1;
}

__device__
inline bool attemptHashedBucketVoxelBlockCreation(size_t & hashedBucketIndex, BlockHeap * blockHeap_d, Vector3f & point_d, HashEntry * hashEntries){
  size_t insertCurrentGlobalIndex = hashedBucketIndex * HASH_ENTRIES_PER_BUCKET;
  for(size_t i=0; i<HASH_ENTRIES_PER_BUCKET; ++i){
    if(hashEntries[insertCurrentGlobalIndex+i].isFree()){ 
      int blockHeapFreeIndex = atomicAdd(&(blockHeap_d->currentIndex), 1);
      VoxelBlock * allocBlock = new VoxelBlock();
      HashEntry * allocBlockHashEntry = new HashEntry(point_d, blockHeapFreeIndex);
      blockHeap_d->blocks[blockHeapFreeIndex] = *allocBlock;
      hashEntries[insertCurrentGlobalIndex+i] = *allocBlockHashEntry;
      hipFree(allocBlock);
      hipFree(allocBlockHashEntry);
      return true;
    }
  }
  return false;
} 

__device__
inline bool attemptLinkedListVoxelBlockCreation(size_t & hashedBucketIndex, BlockHeap * blockHeap_d, HashTable * hashTable_d, size_t & insertBucketIndex, size_t & endLinkedListPointer, Vector3f & point_d, HashEntry * hashEntries){
  size_t insertCurrentGlobalIndex;
  while(insertBucketIndex!=hashedBucketIndex){
    if(!atomicCAS(&hashTable_d->mutex[insertBucketIndex], 0, 1)){
      insertCurrentGlobalIndex = insertBucketIndex * HASH_ENTRIES_PER_BUCKET;
      for(size_t i=0; i<HASH_ENTRIES_PER_BUCKET-1; ++i){
        if(hashEntries[insertCurrentGlobalIndex+i].isFree() ){ 
            int blockHeapFreeIndex = atomicAdd(&(blockHeap_d->currentIndex), 1);
            VoxelBlock * allocBlock = new VoxelBlock();
            HashEntry * allocBlockHashEntry = new HashEntry(point_d, blockHeapFreeIndex);
            blockHeap_d->blocks[blockHeapFreeIndex] = *allocBlock;
            size_t insertPos = insertCurrentGlobalIndex + i;
            hashEntries[insertPos] = *allocBlockHashEntry;
            hipFree(allocBlock);
            hipFree(allocBlockHashEntry);
            if(insertPos > endLinkedListPointer){
              hashEntries[endLinkedListPointer].offset = insertPos - endLinkedListPointer;
            }
            else{
              hashEntries[endLinkedListPointer].offset = HASH_TABLE_SIZE - endLinkedListPointer + insertPos;
            }
            return true;
        }
      }
      atomicExch(&hashTable_d->mutex[insertBucketIndex], 0);
    }
    insertBucketIndex++;
    if(insertBucketIndex == NUM_BUCKETS){
      insertBucketIndex = 0;
    }
  }
  return false;
}
__global__
void allocateVoxelBlocks(Vector3f * points_d, HashTable * hashTable_d, BlockHeap * blockHeap_d, bool * unallocatedPoints_d, int * size_d, int * unallocatedPointsCount_d) //clean up
{
  int threadIndex = (blockIdx.x*threadsPerCudaBlock + threadIdx.x);
  if(threadIndex>=*size_d || (unallocatedPoints_d[threadIndex]==0)){
    return;
  }
  Vector3f point_d = points_d[threadIndex];

  size_t hashedBucketIndex = retrieveHashIndexFromPoint(point_d);
  size_t currentGlobalIndex = hashedBucketIndex * HASH_ENTRIES_PER_BUCKET;
  HashEntry * hashEntries = hashTable_d->hashEntries;

  int block_position = getBlockPositionForBlockCoordinates(point_d, hashedBucketIndex, currentGlobalIndex, hashEntries);

  //block is already allocated
  if(block_position!=-1){
    unallocatedPoints_d[threadIndex] = 0;
    atomicSub(unallocatedPointsCount_d, 1);
    return;
  }

  //attempt to get lock for hashed bucket
  if(!atomicCAS(&hashTable_d->mutex[hashedBucketIndex], 0, 1)){

    if(attemptHashedBucketVoxelBlockCreation(hashedBucketIndex, blockHeap_d, point_d, hashEntries)) {
      unallocatedPoints_d[threadIndex] = 0;
      atomicSub(unallocatedPointsCount_d, 1);
      atomicExch(&hashTable_d->mutex[hashedBucketIndex], 0);
      return;
    }

    size_t insertBucketIndex = hashedBucketIndex + 1;
    if(insertBucketIndex == NUM_BUCKETS){
      insertBucketIndex = 0;
    }

    //current global index will point to end of linked list which includes hashed bucket if no linked list
    size_t endLinkedListBucket = currentGlobalIndex / HASH_ENTRIES_PER_BUCKET;

    bool haveEndLinkedListBucketLock = true;

    if(endLinkedListBucket!=hashedBucketIndex){
      atomicExch(&hashTable_d->mutex[hashedBucketIndex], 0);
      haveEndLinkedListBucketLock = !atomicCAS(&hashTable_d->mutex[endLinkedListBucket], 0, 1);
    }

    if(haveEndLinkedListBucketLock){
      if(attemptLinkedListVoxelBlockCreation(hashedBucketIndex, blockHeap_d, hashTable_d, insertBucketIndex, currentGlobalIndex, point_d, hashEntries)){
        unallocatedPoints_d[threadIndex] = 0;
        atomicSub(unallocatedPointsCount_d, 1); 
        atomicExch(&hashTable_d->mutex[endLinkedListBucket], 0);
        atomicExch(&hashTable_d->mutex[insertBucketIndex], 0);
      }
      else{
        atomicExch(&hashTable_d->mutex[endLinkedListBucket], 0);
      }
      return;
    }
  }
}

__device__
size_t getLocalVoxelIndex(Vector3f diff){
  diff /= VOXEL_SIZE;
  return floor(diff(0)) + (floor(diff(1)) * VOXEL_PER_BLOCK) + (floor(diff(2)) * VOXEL_PER_BLOCK * VOXEL_PER_BLOCK);
}

__device__
float getMagnitude(Vector3f vector){
  return sqrt(pow(vector(0),2) + pow(vector(1),2) + pow(vector(2),2));
}

__device__
float dotProduct(Vector3f a, Vector3f b){
  return a(0)*b(0) + a(1)*b(1) + a(2)*b(2);
}


__device__
float getDistanceUpdate(Vector3f voxelCoordinates, Vector3f point_d, Vector3f origin){
//x:center of current voxel   p:position of lidar point   s:sensor origin
Vector3f lidarOriginDiff = point_d - origin; //p-s
Vector3f lidarVoxelDiff = point_d - voxelCoordinates; //p-x
float magnitudeLidarVoxelDiff = getMagnitude(lidarVoxelDiff);
float dotProd = dotProduct(lidarOriginDiff, lidarVoxelDiff);
if(dotProd < 0){
  magnitudeLidarVoxelDiff *=-1;
}

return magnitudeLidarVoxelDiff;
}

__global__
void updateVoxels(Vector3f * voxels, HashTable * hashTable_d, BlockHeap * blockHeap_d, Vector3f * point_d, Vector3f * origin, int * size){
  int threadIndex = blockIdx.x*threadsPerCudaBlock + threadIdx.x;
  if(threadIndex >= * size){
    return;
  }
  Vector3f voxelCoordinates = voxels[threadIndex];
  Vector3f voxelBlockCoordinates = GetVoxelBlockCenterFromPoint(voxelCoordinates);

  size_t bucketIndex = retrieveHashIndexFromPoint(voxelBlockCoordinates);
  size_t currentGlobalIndex = bucketIndex * HASH_ENTRIES_PER_BUCKET;
  HashEntry * hashEntries = hashTable_d->hashEntries;

  //make a global vector with half voxel block size values
  int voxelBlockHeapPosition = getBlockPositionForBlockCoordinates(voxelBlockCoordinates, bucketIndex, currentGlobalIndex, hashEntries);
  Vector3f voxelBlockBottomLeftCoordinates;
  voxelBlockBottomLeftCoordinates(0) = voxelBlockCoordinates(0)-HALF_VOXEL_BLOCK_SIZE;
  voxelBlockBottomLeftCoordinates(1) = voxelBlockCoordinates(1)-HALF_VOXEL_BLOCK_SIZE;
  voxelBlockBottomLeftCoordinates(2) = voxelBlockCoordinates(2)-HALF_VOXEL_BLOCK_SIZE;
  size_t localVoxelIndex = getLocalVoxelIndex(voxelCoordinates - voxelBlockBottomLeftCoordinates);
  Vector3f point = * point_d;
  VoxelBlock * block = &(blockHeap_d->blocks[voxelBlockHeapPosition]);
  Voxel* voxel = &(block->voxels[localVoxelIndex]);
  int * mutex = &(block->mutex[localVoxelIndex]);

  float weight = 1;
  float distance = getDistanceUpdate(voxelCoordinates, *point_d, *origin);
  float weightTimesDistance = weight * distance;

  //get lock for voxel
  bool updatedVoxel = false;
  while(!updatedVoxel){
    if(!atomicCAS(mutex, 0, 1)){
      updatedVoxel = true;
      //update sdf and weight
      float oldWeight = voxel->weight;
      float oldSdf = voxel->sdf;
      float newWeight = oldWeight + weight;
      float newDistance = (oldWeight * oldSdf + weightTimesDistance) / newWeight;
      voxel->sdf = newDistance;
      newWeight = min(newWeight, MAX_WEIGHT);
      voxel->weight = newWeight;
      // printf("voxel coords: (%f, %f, %f) with sdf: %f with weight: %f\n", voxelCoordinates(0),voxelCoordinates(1), voxelCoordinates(2), voxel->sdf, voxel->weight);
      atomicExch(mutex, 0);
    }
  }
}

__global__
void getVoxelsForPoint(pcl::PointXYZ * points_d, Vector3f * origin_transformed_d, HashTable * hashTable_d, BlockHeap * blockHeap_d, int * size_d){
  int threadIndex = (blockIdx.x*threadsPerCudaBlock + threadIdx.x);
  if(threadIndex>=*size_d){
  return;
  }
  pcl::PointXYZ point_d = points_d[threadIndex];
  Vector3f truncation_start;
  Vector3f truncation_end;
  Vector3f u;
  Vector3f v;

  getTruncationLineEndPoints(point_d, origin_transformed_d, truncation_start, truncation_end, u, v);

  Vector3f truncation_start_voxel = GetVoxelCenterFromPoint(truncation_start);
  Vector3f truncation_end_voxel = GetVoxelCenterFromPoint(truncation_end);

  Vector3f * voxels = new Vector3f[200]; //set in terms of truncation distance and voxel size
  int * size = new int(0);

  traverseVolume(truncation_start_voxel, truncation_end_voxel, VOXEL_SIZE, u, v, voxels, size);

  Vector3f * lidarPoint = new Vector3f(point_d.x, point_d.y, point_d.z);

  int numCudaBlocks = *size/threadsPerCudaBlock + 1;
  updateVoxels<<<numCudaBlocks, threadsPerCudaBlock>>>(voxels, hashTable_d, blockHeap_d, lidarPoint, origin_transformed_d, size);
  cdpErrchk(hipPeekAtLastError());
  hipDeviceSynchronize();

  hipFree(lidarPoint);
  hipFree(voxels);
  hipFree(size);
  return;
}

__global__
void processOccupiedVoxelBlock(Vector3f * occupiedVoxels, int * index, Voxel * sdfWeightVoxelVals_d, Vector3f * position, VoxelBlock * block){
  int threadIndex = blockIdx.x*threadsPerCudaBlock + threadIdx.x;
  if(threadIndex >= VOXEL_PER_BLOCK * VOXEL_PER_BLOCK * VOXEL_PER_BLOCK){
    return;
  }

  int voxelIndex = threadIndex;
  Voxel voxel = block->voxels[threadIndex];
  if(voxel.weight!=0){
    float z = voxelIndex / (VOXEL_PER_BLOCK * VOXEL_PER_BLOCK);
    voxelIndex -= z*VOXEL_PER_BLOCK*VOXEL_PER_BLOCK;
    float y = voxelIndex / VOXEL_PER_BLOCK;
    voxelIndex -= y*VOXEL_PER_BLOCK;
    float x = voxelIndex;

    Vector3f positionVec = * position;
    float xCoord = x * VOXEL_SIZE + HALF_VOXEL_SIZE + positionVec(0);
    float yCoord = y * VOXEL_SIZE + HALF_VOXEL_SIZE + positionVec(1);
    float zCoord = z * VOXEL_SIZE + HALF_VOXEL_SIZE + positionVec(2);
  
    Vector3f v(xCoord, yCoord, zCoord);
    int occupiedVoxelIndex = atomicAdd(&(*index), 1);
    if(occupiedVoxelIndex<OCCUPIED_VOXELS_SIZE){
      occupiedVoxels[occupiedVoxelIndex] = v;
      sdfWeightVoxelVals_d[occupiedVoxelIndex] = voxel;
    }
  }
}

__global__
void visualizeOccupiedVoxels(HashTable * hashTable_d, BlockHeap * blockHeap_d, Vector3f * occupiedVoxels, int * index, Voxel * sdfWeightVoxelVals_d){
  int threadIndex = blockIdx.x*threadsPerCudaBlock +threadIdx.x;
  if(threadIndex >= HASH_TABLE_SIZE) return;
  HashEntry hashEntry = hashTable_d->hashEntries[threadIndex];
  if(hashEntry.isFree()){
    return;
  }
  int pointer = hashEntry.pointer;
  Vector3f * position = new Vector3f(hashEntry.position(0) - HALF_VOXEL_BLOCK_SIZE, 
  hashEntry.position(1)- HALF_VOXEL_BLOCK_SIZE,
  hashEntry.position(2)- HALF_VOXEL_BLOCK_SIZE);

  VoxelBlock * block = &(blockHeap_d->blocks[pointer]);
  int size = VOXEL_PER_BLOCK * VOXEL_PER_BLOCK * VOXEL_PER_BLOCK;
  int numBlocks = size/threadsPerCudaBlock + 1;
  processOccupiedVoxelBlock<<<numBlocks,threadsPerCudaBlock>>>(occupiedVoxels, index, sdfWeightVoxelVals_d, position, block);
  cdpErrchk(hipPeekAtLastError());
  hipFree(position);
}

TSDFHandler::TSDFHandler(){
  tsdfContainer = new TSDFContainer();
}

TSDFHandler::~TSDFHandler(){
  free(tsdfContainer);
}

void TSDFHandler::processPointCloudAndUpdateVoxels(pcl::PointCloud<pcl::PointXYZ>::Ptr pointcloud, Vector3f * origin_transformed_h, Vector3f * occupied_voxels_h, int * occupied_voxels_index, Voxel * sdfWeightVoxelVals_h)
{ 
  std::vector<pcl::PointXYZ, Eigen::aligned_allocator<pcl::PointXYZ>> points = pointcloud->points;

  pcl::PointXYZ * points_h = &points[0];
  pcl::PointXYZ * points_d;
  int pointcloud_size = pointcloud->size();
  int * pointcloud_size_d;

  hipMalloc(&points_d, sizeof(*points_h)*pointcloud_size);
  hipMemcpy(points_d, points_h, sizeof(*points_h)*pointcloud_size, hipMemcpyHostToDevice);
  hipMalloc(&pointcloud_size_d, sizeof(int));
  hipMemcpy(pointcloud_size_d, &pointcloud_size, sizeof(int), hipMemcpyHostToDevice);

  Vector3f * origin_transformed_d;
  hipMalloc(&origin_transformed_d, sizeof(*origin_transformed_h));
  hipMemcpy(origin_transformed_d, origin_transformed_h,sizeof(*origin_transformed_h),hipMemcpyHostToDevice);

  HashTable * hash_table_d = tsdfContainer->getCudaHashTable();
  BlockHeap * block_heap_d = tsdfContainer->getCudaBlockHeap();

  allocateVoxelBlocksAndUpdateVoxels(points_d, origin_transformed_d, pointcloud_size_d, pointcloud_size, hash_table_d, block_heap_d);

  visualize(occupied_voxels_h, occupied_voxels_index, sdfWeightVoxelVals_h, hash_table_d, block_heap_d);

  hipFree(pointcloud_size_d);
  hipFree(points_d);
  hipFree(origin_transformed_d);

}

void TSDFHandler::allocateVoxelBlocksAndUpdateVoxels(pcl::PointXYZ * points_d, Vector3f * origin_transformed_d, int * pointcloud_size_d, int pointcloud_size, HashTable * hash_table_d, BlockHeap * block_heap_d){
    //TODO: FIX
  // int maxBlocksPerPoint = ceil(pow(truncation_distance,3) / pow(VOXEL_BLOCK_SIZE, 3));
  int maxBlocks = 10 * pointcloud_size;
  Vector3f pointcloud_voxel_blocks_h[maxBlocks];
  Vector3f * pointcloud_voxel_blocks_d;
  int * pointcloud_voxel_blocks_h_index = new int(0);
  int * pointcloud_voxel_blocks_d_index;
  hipMalloc(&pointcloud_voxel_blocks_d, sizeof(*pointcloud_voxel_blocks_h)*maxBlocks);
  hipMemcpy(pointcloud_voxel_blocks_d, pointcloud_voxel_blocks_h, sizeof(*pointcloud_voxel_blocks_h)*maxBlocks,hipMemcpyHostToDevice); //do I even need to memcpy
  hipMalloc(&pointcloud_voxel_blocks_d_index, sizeof(*pointcloud_voxel_blocks_h_index));
  hipMemcpy(pointcloud_voxel_blocks_d_index, pointcloud_voxel_blocks_h_index, sizeof(*pointcloud_voxel_blocks_h_index), hipMemcpyHostToDevice);

    //since size can go over threads per block allocate this properly to include all data
  int num_cuda_blocks = pointcloud_size / threadsPerCudaBlock + 1;

  getVoxelBlocks(num_cuda_blocks, points_d, pointcloud_voxel_blocks_d, pointcloud_voxel_blocks_d_index, origin_transformed_d, pointcloud_size_d);

  integrateVoxelBlockPointsIntoHashTable(pointcloud_voxel_blocks_d, pointcloud_voxel_blocks_d_index, hash_table_d, block_heap_d);

  updateVoxels(num_cuda_blocks, points_d, origin_transformed_d, pointcloud_size_d, hash_table_d, block_heap_d);

  hipFree(pointcloud_voxel_blocks_d);
  hipFree(pointcloud_voxel_blocks_d_index);
  free(pointcloud_voxel_blocks_h_index);

}

void TSDFHandler::getVoxelBlocks(int num_cuda_blocks, pcl::PointXYZ * points_d, Vector3f * pointcloud_voxel_blocks_d, int * pointcloud_voxel_blocks_d_index, Vector3f * origin_transformed_d, int * pointcloud_size_d){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  getVoxelBlocksForPoint<<<num_cuda_blocks,threadsPerCudaBlock>>>(points_d, pointcloud_voxel_blocks_d, pointcloud_voxel_blocks_d_index, origin_transformed_d, pointcloud_size_d);
  gpuErrchk( hipPeekAtLastError() );
  hipDeviceSynchronize();

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Get Voxel Block Duration: %f\n", milliseconds);
}

void TSDFHandler::integrateVoxelBlockPointsIntoHashTable(Vector3f * points_d, int * pointcloud_voxel_blocks_d_index, HashTable * hash_table_d, BlockHeap * block_heap_d){

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  int * size_h = new int(0);
  hipMemcpy(size_h, pointcloud_voxel_blocks_d_index, sizeof(int), hipMemcpyDeviceToHost);
  int size = * size_h;

  bool * unallocatedPoints_h = new bool[size];
  for(int i=0;i<size;++i)
  {
    unallocatedPoints_h[i] = 1;
  }
  bool * unallocatedPoints_d;
  hipMalloc(&unallocatedPoints_d, sizeof(*unallocatedPoints_h)*size);
  hipMemcpy(unallocatedPoints_d, unallocatedPoints_h, sizeof(*unallocatedPoints_h)*size, hipMemcpyHostToDevice);

  int * unallocatedPointsCount_h = new int(size);
  int * unallocatedPointsCount_d;
  hipMalloc(&unallocatedPointsCount_d, sizeof(*unallocatedPointsCount_h));
  hipMemcpy(unallocatedPointsCount_d, unallocatedPointsCount_h, sizeof(*unallocatedPointsCount_h), hipMemcpyHostToDevice);

  int num_cuda_blocks = size / threadsPerCudaBlock + 1;
  while(*unallocatedPointsCount_h > 0){ //POSSIBILITY OF INFINITE LOOP if no applicable space is left for an unallocated block even if there is still space left in hash table
    allocateVoxelBlocks<<<num_cuda_blocks,threadsPerCudaBlock>>>(points_d, hash_table_d, block_heap_d, unallocatedPoints_d, pointcloud_voxel_blocks_d_index, unallocatedPointsCount_d);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();
    hipMemcpy(unallocatedPointsCount_h, unallocatedPointsCount_d, sizeof(*unallocatedPointsCount_h), hipMemcpyDeviceToHost);
  }

  printHashTableAndBlockHeap<<<1,1>>>(hash_table_d, block_heap_d);
  hipDeviceSynchronize();

  hipFree(unallocatedPoints_d);
  hipFree(unallocatedPointsCount_d);
  delete size_h;
  delete unallocatedPoints_h;
  delete unallocatedPointsCount_h;

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Integrate Voxel Block Duration: %f\n", milliseconds);
}

void TSDFHandler::updateVoxels(int & num_cuda_blocks, pcl::PointXYZ * points_d, Vector3f * origin_transformed_d, int * pointcloud_size_d, HashTable * hash_table_d, BlockHeap * block_heap_d){

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  getVoxelsForPoint<<<num_cuda_blocks,threadsPerCudaBlock>>>(points_d, origin_transformed_d, hash_table_d, block_heap_d, pointcloud_size_d);
  gpuErrchk( hipPeekAtLastError() );
  hipDeviceSynchronize();

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Update Voxels Duration: %f\n", milliseconds);
}

void TSDFHandler::visualize(Vector3f * occupied_voxels_h, int * occupied_voxels_index, Voxel * sdfWeightVoxelVals_h, HashTable * hash_table_d, BlockHeap * block_heap_d){
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  Vector3f * occupied_voxels_d;
  int * occupied_voxels_index_d;
  Voxel * sdfWeightVoxelVals_d;
  int occupiedVoxelsSize = OCCUPIED_VOXELS_SIZE;
  hipMalloc(&occupied_voxels_d, sizeof(*occupied_voxels_h)*occupiedVoxelsSize);
  hipMemcpy(occupied_voxels_d, occupied_voxels_h, sizeof(*occupied_voxels_h)*occupiedVoxelsSize,hipMemcpyHostToDevice);
  hipMalloc(&occupied_voxels_index_d, sizeof(*occupied_voxels_index));
  hipMemcpy(occupied_voxels_index_d, occupied_voxels_index, sizeof(*occupied_voxels_index), hipMemcpyHostToDevice);
  hipMalloc(&sdfWeightVoxelVals_d, sizeof(*sdfWeightVoxelVals_h)*occupiedVoxelsSize);
  hipMemcpy(sdfWeightVoxelVals_d, sdfWeightVoxelVals_h, sizeof(*sdfWeightVoxelVals_h)*occupiedVoxelsSize, hipMemcpyHostToDevice);

  int numVisVoxBlocks = HASH_TABLE_SIZE / threadsPerCudaBlock + 1;
  visualizeOccupiedVoxels<<<numVisVoxBlocks,threadsPerCudaBlock>>>(hash_table_d, block_heap_d, occupied_voxels_d, occupied_voxels_index_d, sdfWeightVoxelVals_d);
  gpuErrchk( hipPeekAtLastError() );
  hipDeviceSynchronize();

  hipMemcpy(occupied_voxels_h, occupied_voxels_d, sizeof(*occupied_voxels_h)*occupiedVoxelsSize, hipMemcpyDeviceToHost);
  hipMemcpy(occupied_voxels_index, occupied_voxels_index_d, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(sdfWeightVoxelVals_h, sdfWeightVoxelVals_d, sizeof(*sdfWeightVoxelVals_h)*occupiedVoxelsSize, hipMemcpyDeviceToHost);

  hipFree(occupied_voxels_d); //instead of allocating and freeing over and over just add to tsdfhandler
  hipFree(occupied_voxels_index_d);
  hipFree(sdfWeightVoxelVals_d);

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("Visualize Voxels Duration: %f\n", milliseconds);

}