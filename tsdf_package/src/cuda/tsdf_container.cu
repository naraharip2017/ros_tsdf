#include <stdio.h>

#include "cuda/tsdf_container.cuh"

/*
* Definitions for tsdf_container class declared in tsdf_container.cuh
*/

TSDFContainer::TSDFContainer(){
    //allocate hash table and block heap on host and device
    hashTable_h = new HashTable();
    blockHeap_h = new BlockHeap();
    hipMalloc(&hashTable_d, sizeof(*hashTable_h));
    hipMemcpy(hashTable_d, hashTable_h, sizeof(*hashTable_h), hipMemcpyHostToDevice);
    hipMalloc(&blockHeap_d, sizeof(*blockHeap_h));
    hipMemcpy(blockHeap_d, blockHeap_h, sizeof(*blockHeap_h), hipMemcpyHostToDevice);
}

TSDFContainer::~TSDFContainer(){
    hipFree(hashTable_d);
    hipFree(blockHeap_d);
    delete hashTable_h;
    delete blockHeap_h;
}

HashTable * TSDFContainer::getCudaHashTable(){
    return hashTable_d;
}

BlockHeap * TSDFContainer::getCudaBlockHeap(){
    return blockHeap_d;
}
