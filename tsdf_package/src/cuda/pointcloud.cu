#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "pointcloud.cuh"
#include "tsdf_handler.cuh"
// #include <memory>
// #include <cstdio>
#include <pcl/point_types.h>
#include <pcl/PCLPointCloud2.h>
#include <pcl/conversions.h>
#include <math.h>

typedef Eigen::Matrix<float, 3, 1> Vector3f;

__constant__
const float truncation_distance = 2.0;

//rename to pointcloud_handler

__global__
void printVoxelBlocksFromPoint(Vector3f * pointCloudVoxelBlocks_d, int * pointer_d){
  printf("List of Points: \n");
  for(int i=0;i<*pointer_d;++i){
    Vector3f point = pointCloudVoxelBlocks_d[i];
    printf("(%f, %f, %f)\n", point(0), point(1), point(2));
  }
}

__device__
size_t retrieveHash(Vector3f point){ //tested using int can get negatives
  return abs((((int)point(0)*PRIME_ONE) ^ ((int)point(1)*PRIME_TWO) ^ ((int)point(2)*PRIME_THREE)) % NUM_BUCKETS);
}

__device__ 
float FloorFun(float x, float scale){
  return floor(x*scale) / scale;
}

__device__
Vector3f GetVoxelBlockCenterFromPoint(Vector3f point){
  float scale = 1 / VOXEL_BLOCK_SIZE;
  Vector3f blockCenter;
  blockCenter(0) = FloorFun(point(0), scale) + HALF_VOXEL_BLOCK_SIZE;
  blockCenter(1) = FloorFun(point(1), scale) + HALF_VOXEL_BLOCK_SIZE;
  blockCenter(2) = FloorFun(point(2), scale) + HALF_VOXEL_BLOCK_SIZE;
  return blockCenter;
}

__device__
bool checkFloatingPointVectorsEqual(Vector3f A, Vector3f B){
  Vector3f diff = A-B;
  if((fabs(diff(0)) < EPSILON) && (fabs(diff(1)) < EPSILON) && (fabs(diff(2)) < EPSILON))
    return true;

  return false;
}

 __global__
 void getVoxelBlocksForPoint(pcl::PointXYZ * points_d, Vector3f * pointCloudVoxelBlocks_d, int * pointer_d, Vector3f * origin_transformed_d){
  int threadIndex = threadIdx.x;
  pcl::PointXYZ point_d = points_d[threadIndex];
  // Point * origin = new Point(0,0,0); //make these vectors?
  // Point * Point_points_d = new Point(point_d.x, point_d.y, point_d.z); //converts the float to int
  // Point * direction = *Point_points_d - *origin;
  Vector3f u = *origin_transformed_d;
  printf("transformation: (%f, %f, %f)\n", u(0), u(1), u(2));
  Vector3f point_d_vector(point_d.x, point_d.y, point_d.z);
  Vector3f v = point_d_vector - u; //direction
  printf("V: (%f, %f, %f)\n", v(0), v(1), v(2));
  //equation of line is u+tv
  float vMag = sqrt(pow(v(0), 2) + pow(v(1),2) + pow(v(2), 2));
  Vector3f v_normalized = v / vMag;
  Vector3f truncation_start = point_d_vector - truncation_distance*v_normalized;
  printf("Truncation start : (%f, %f, %f)\n", truncation_start(0), truncation_start(1), truncation_start(2));
  
  Vector3f truncation_end = point_d_vector + truncation_distance*v_normalized;  //get voxel block of this and then traverse through voxel blocks till it equals this one
  printf("Truncation end : (%f, %f, %f)\n", truncation_end(0), truncation_end(1), truncation_end(2));

  float distance_tStart_origin = sqrt(pow(truncation_start(0) - u(0), 2) + pow(truncation_start(1) - u(1),2) + pow(truncation_start(2) - u(2), 2));
  float distance_tEnd_origin = sqrt(pow(truncation_end(0) - u(0), 2) + pow(truncation_end(1) - u(1),2) + pow(truncation_end(2) - u(2), 2));

  if(distance_tEnd_origin < distance_tStart_origin){
    Vector3f temp = truncation_start;
    truncation_start = truncation_end;
    truncation_end = temp;
  }

  Vector3f truncation_start_block = GetVoxelBlockCenterFromPoint(truncation_start);
  printf("Truncation start Block: (%f, %f, %f), hashes to %lu\n", truncation_start_block(0), truncation_start_block(1), truncation_start_block(2), retrieveHash(truncation_start_block));
  // printf("point in size_t: %d, %d, %d\n", (int)truncation_start_block(0), (int)truncation_start_block(1), (int)truncation_start_block(2));
  Vector3f truncation_end_block = GetVoxelBlockCenterFromPoint(truncation_end);
  printf("Truncation end Block: (%f, %f, %f), hashes to %lu\n", truncation_end_block(0), truncation_end_block(1), truncation_end_block(2), retrieveHash(truncation_end_block));
  // printf("point in size_t: %d, %d, %d\n", (int)truncation_end_block(0), (int)truncation_end_block(1), (int)truncation_end_block(2));
  float stepX = v(0) > 0 ? VOXEL_BLOCK_SIZE : -1 * VOXEL_BLOCK_SIZE;
  float stepY = v(1) > 0 ? VOXEL_BLOCK_SIZE : -1 * VOXEL_BLOCK_SIZE;
  float stepZ = v(2) > 0 ? VOXEL_BLOCK_SIZE : -1 * VOXEL_BLOCK_SIZE;
  float tMaxX = fabs(v(0) < 0 ? (truncation_start_block(0) - HALF_VOXEL_BLOCK_SIZE - u(0)) / v(0) : (truncation_start_block(0) + HALF_VOXEL_BLOCK_SIZE - u(0)) / v(0));
  float tMaxY = fabs(v(1) < 0 ? (truncation_start_block(1) - HALF_VOXEL_BLOCK_SIZE - u(1)) / v(1) : (truncation_start_block(1) + HALF_VOXEL_BLOCK_SIZE - u(1)) / v(1));
  float tMaxZ = fabs(v(2) < 0 ? (truncation_start_block(2) - HALF_VOXEL_BLOCK_SIZE - u(2)) / v(2) : (truncation_start_block(2) + HALF_VOXEL_BLOCK_SIZE - u(2)) / v(2));
  float tDeltaX = fabs(VOXEL_BLOCK_SIZE / v(0));
  float tDeltaY = fabs(VOXEL_BLOCK_SIZE / v(1));
  float tDeltaZ = fabs(VOXEL_BLOCK_SIZE / v(2));
  Vector3f currentBlock(truncation_start_block(0), truncation_start_block(1), truncation_start_block(2));

  do{
    //add current block to blocks in current frame list or whatever
    int pointCloudVoxelBlocksIndex = atomicAdd(&(*pointer_d), 1);
    pointCloudVoxelBlocks_d[pointCloudVoxelBlocksIndex] = currentBlock;
    printf("Current Block: (%f, %f, %f), hashes to %lu\n", currentBlock(0), currentBlock(1), currentBlock(2), retrieveHash
    (currentBlock));
    // printf("point in size_t: %d, %d, %d\n", (int)currentBlock(0), (int)currentBlock(1), (int)currentBlock(2));
    if(tMaxX < tMaxY){
      if(tMaxX < tMaxZ)
      {
        currentBlock(0) += stepX;
        tMaxX += tDeltaX;
      }
      else if(tMaxX > tMaxZ){
        currentBlock(2) += stepZ;
        tMaxZ += tDeltaZ;
      }
      else{
        currentBlock(0) += stepX;
        currentBlock(2) += stepZ;
        tMaxX += tDeltaX;
        tMaxZ += tDeltaZ;
      }
    }
    else if(tMaxX > tMaxY){
      if(tMaxY < tMaxZ){
        currentBlock(1) += stepY;
        tMaxY += tDeltaY;
      }
      else if(tMaxY > tMaxZ){
        currentBlock(2) += stepZ;
        tMaxZ += tDeltaZ;
      }
      else{
        currentBlock(1) += stepY;
        currentBlock(2) += stepZ;
        tMaxY += tDeltaY;
        tMaxZ += tDeltaZ;
      }
    }
    else{
      if(tMaxZ < tMaxX){
        currentBlock(2) += stepZ;
        tMaxZ += tDeltaZ;
      }
      else if(tMaxZ > tMaxX){
        currentBlock(0) += stepX;
        currentBlock(1) += stepY;
        tMaxX += tDeltaX;
        tMaxY += tDeltaY;
      }
      else{ //can remove equals statements if want to improve on performance
        currentBlock(0) += stepX;
        currentBlock(1) += stepY;
        currentBlock(2) += stepZ;
        tMaxX += tDeltaX;
        tMaxY += tDeltaY;
        tMaxZ += tDeltaZ;
      }
    }       
  } while(!checkFloatingPointVectorsEqual(currentBlock, truncation_end_block));
  int pointCloudVoxelBlocksIndex = atomicAdd(&(*pointer_d), 1);
  pointCloudVoxelBlocks_d[pointCloudVoxelBlocksIndex] = currentBlock;
  printf("Current Block: (%f, %f, %f), hashes to %lu\n", currentBlock(0), currentBlock(1), currentBlock(2), retrieveHash(currentBlock));
  // printf("point in size_t: %d, %d, %d\n", (int)currentBlock(0), (int)currentBlock(1), (int)currentBlock(2));
  printf("Cloud with Points: %f, %f, %f\n", points_d[threadIndex].x,points_d[threadIndex].y,points_d[threadIndex].z);
  return;
 }


//takes sensor origin position
void pointcloudMain(pcl::PointCloud<pcl::PointXYZ>::Ptr pointcloud, Vector3f * origin_transformed_h)
{
  //retrieve sensor origin..can use transformation from point cloud time stamp drone_1/lidar frame to drone_1 frame then transform 0,0,0
  
  std::vector<pcl::PointXYZ, Eigen::aligned_allocator<pcl::PointXYZ>> points = pointcloud->points;

  pcl::PointXYZ * points_h = &points[0];
  pcl::PointXYZ * points_d;
  int size = pointcloud->size();
  hipMalloc(&points_d, sizeof(*points_h)*size);
  hipMemcpy(points_d, points_h, sizeof(*points_h)*size, hipMemcpyHostToDevice);

  // int * pointCloudVoxelBlockSize_h = new int(maxBlocksPerPoint * size);
  // int * pointCloudVoxelBlockSize_d;
  // hipMalloc(&pointCloudVoxelBlockSize_d, sizeof(int));
  // hipMemcpy(pointCloudVoxelBlockSize_d, pointCloudVoxelBlockSize_h, sizeof(int, ))
  int maxBlocksPerPoint = ceil(pow(truncation_distance,3) / pow(VOXEL_BLOCK_SIZE, 3));
  int maxBlocks = maxBlocksPerPoint * size;
  Vector3f * pointCloudVoxelBlocks_h[maxBlocks];
  Vector3f * pointCloudVoxelBlocks_d;
  int * pointer_h = 0;
  int * pointer_d;
  hipMalloc(&pointCloudVoxelBlocks_d, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks);
  hipMemcpy(pointCloudVoxelBlocks_d, pointCloudVoxelBlocks_h, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks,hipMemcpyHostToDevice); //do I even need to memcpy
  hipMalloc(&pointer_d, sizeof(*pointer_h));
  hipMemcpy(pointer_d, pointer_h, sizeof(*pointer_h), hipMemcpyHostToDevice);

  Vector3f * origin_transformed_d;
  hipMalloc(&origin_transformed_d, sizeof(*origin_transformed_h));
  hipMemcpy(origin_transformed_d, origin_transformed_h,sizeof(*origin_transformed_h),hipMemcpyHostToDevice);
  // PointCloudVoxelBlocks * pointCloudVoxelBlocks_h = new PointCloudVoxelBlocks(maxBlocks);
  // PointCloudVoxelBlocks * pointCloudVoxelBlocks_d;

  // hipMalloc(&pointCloudVoxelBlocks_d, sizeof(float)*3*maxBlocks+4);
  // hipMemcpy(pointCloudVoxelBlocks_d, pointCloudVoxelBlocks_h, sizeof(Vector3f)*maxBlocks+sizeof(int), hipMemcpyHostToDevice);

  getVoxelBlocksForPoint<<<1,size>>>(points_d, pointCloudVoxelBlocks_d, pointer_d, origin_transformed_d);


  // for(size_t i=0; i<points.size(); ++i){
  //     printf("Cloud with Points: %f, %f, %f\n", points[i].x,points[i].y,points[i].z);
  //   } 
}

void testVoxelBlockTraversal(){
  // float f = 10.23423;
  int size = 2;
  pcl::PointXYZ * point1 = new pcl::PointXYZ(-73.4567,33.576, 632.8910);
  pcl::PointXYZ * point2 = new pcl::PointXYZ(-7.23421,-278, 576.2342);
  pcl::PointXYZ * points_h = new pcl::PointXYZ[size];
  points_h[0] = *point1;
  points_h[1] = *point2;
  pcl::PointXYZ * points_d;
  hipMalloc(&points_d, sizeof(*points_h)*size);
  hipMemcpy(points_d, points_h, sizeof(*points_h)*size, hipMemcpyHostToDevice);


  int maxBlocksPerPoint = ceil(pow(truncation_distance,3) / pow(VOXEL_BLOCK_SIZE, 3));
  int maxBlocks = maxBlocksPerPoint * size;
  Vector3f * pointCloudVoxelBlocks_h[maxBlocks]; //make these member functions of tsdf_handler if cant pass device reference on host code
  Vector3f * pointCloudVoxelBlocks_d;
  int * pointer_h = 0;
  int * pointer_d;
  hipMalloc(&pointCloudVoxelBlocks_d, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks);
  hipMemcpy(pointCloudVoxelBlocks_d, pointCloudVoxelBlocks_h, sizeof(*pointCloudVoxelBlocks_h)*maxBlocks,hipMemcpyHostToDevice); //do I even need to memcpy
  hipMalloc(&pointer_d, sizeof(*pointer_h));
  hipMemcpy(pointer_d, pointer_h, sizeof(*pointer_h), hipMemcpyHostToDevice);

  // getVoxelBlocksForPoint<<<1,size>>>(points_d, pointCloudVoxelBlocks_d, pointer_d);

  printVoxelBlocksFromPoint<<<1,1>>>(pointCloudVoxelBlocks_d, pointer_d);

  hipDeviceSynchronize();
}